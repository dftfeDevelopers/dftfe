// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2022 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das
//

// skip1 and skip2 are flags used by chebyshevFilter function to perform overlap
// of computation and communication. When either skip1 or skip2 flags are set to
// true all communication calls are skipped as they are directly called in
// chebyshevFilter Only one of the skip flags is set to true in a call. When
// skip1 is set to true extraction and C^{T}*X computation are skipped and
// computations directly start from V*C^{T}*X. When skip2 is set to true only
// extraction and C^{T}*X computations are performed.
template <unsigned int FEOrder, unsigned int FEOrderElectro>
void
kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
  computeNonLocalHamiltonianTimesX(
    const dataTypes::numberGPU *             src,
    distributedGPUVec<dataTypes::numberGPU> &projectorKetTimesVector,
    const unsigned int                       numberWaveFunctions,
    dataTypes::numberGPU *                   dst,
    const bool                               skip1,
    const bool                               skip2)
{
  const dataTypes::number scalarCoeffAlpha = dataTypes::number(1.0),
                          scalarCoeffBeta  = dataTypes::number(0.0);

  //
  // compute C^{\dagger}*X
  //
  unsigned int strideA = numberWaveFunctions * d_numberNodesPerElement;
  unsigned int strideB = d_numberNodesPerElement * d_maxSingleAtomPseudoWfc;
  unsigned int strideC = numberWaveFunctions * d_maxSingleAtomPseudoWfc;

  if (d_totalNonlocalElems > 0 && !skip1)
    {
      cublasXgemmBatched(
        d_cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numberWaveFunctions,
        d_maxSingleAtomPseudoWfc,
        d_numberNodesPerElement,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
        (const dataTypes::numberGPU **)d_A,
        numberWaveFunctions,
        (const dataTypes::numberGPU **)d_B,
        d_numberNodesPerElement,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
        d_C,
        numberWaveFunctions,
        d_totalNonlocalElems);

      cublasXgemm(
        d_cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numberWaveFunctions,
        d_totalPseudoWfcNonLocal,
        d_totalNonlocalElems * d_maxSingleAtomPseudoWfc,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
        reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsDevice[0])),
        numberWaveFunctions,
        reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsReductionDevice[0])),
        d_totalNonlocalElems * d_maxSingleAtomPseudoWfc,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
        reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorParFlattenedDevice[0])),
        numberWaveFunctions);
    }

  // this routine was interfering with overlapping communication and compute. So
  // called separately inside chebyshevFilter. So skip this if either skip1 or
  // skip2 are set to true
  if (!skip1 && !skip2)
    projectorKetTimesVector.setZero();


  if (d_totalNonlocalElems > 0 && !skip1)
    copyToDealiiParallelNonLocalVec<<<(numberWaveFunctions + 255) / 256 *
                                        d_totalPseudoWfcNonLocal,
                                      256>>>(
      numberWaveFunctions,
      d_totalPseudoWfcNonLocal,
      reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
        &d_projectorKetTimesVectorParFlattenedDevice[0])),
      projectorKetTimesVector.begin(),
      thrust::raw_pointer_cast(&d_projectorIdsParallelNumberingMapDevice[0]));

  // Operations related to skip2 (extraction and C^{T}*X) are over. So return
  // control back to chebyshevFilter
  if (skip2)
    return;

  if (!skip1)
    {
      projectorKetTimesVector.compressAdd();
      projectorKetTimesVector.updateGhostValues();
    }

  //
  // Start operations related to skip1 (V*C^{\dagger}*X, C*V*C^{\dagger}*X and
  // assembly)
  //
  if (d_totalNonlocalElems > 0)
    {
      //
      // compute V*C^{\dagger}*X
      //
      scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 *
                          d_totalPseudoWfcNonLocal,
                        256>>>(numberWaveFunctions,
                               d_totalPseudoWfcNonLocal,
                               1.0,
                               projectorKetTimesVector.begin(),
                               thrust::raw_pointer_cast(
                                 &d_nonLocalPseudoPotentialConstantsDevice[0]));

      copyFromParallelNonLocalVecToAllCellsVec<<<(numberWaveFunctions + 255) /
                                                   256 * d_totalNonlocalElems *
                                                   d_maxSingleAtomPseudoWfc,
                                                 256>>>(
        numberWaveFunctions,
        d_totalNonlocalElems,
        d_maxSingleAtomPseudoWfc,
        projectorKetTimesVector.begin(),
        reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsDevice[0])),
        thrust::raw_pointer_cast(
          &d_indexMapFromPaddedNonLocalVecToParallelNonLocalVecDevice[0]));

      //
      // compute C*V*C^{\dagger}*x
      //

      strideA = numberWaveFunctions * d_maxSingleAtomPseudoWfc;
      strideB = d_maxSingleAtomPseudoWfc * d_numberNodesPerElement;
      strideC = numberWaveFunctions * d_numberNodesPerElement;
      cublasXgemmStridedBatched(
        d_cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numberWaveFunctions,
        d_numberNodesPerElement,
        d_maxSingleAtomPseudoWfc,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
        reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsDevice[0])),
        numberWaveFunctions,
        strideA,
        reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_cellHamiltonianMatrixNonLocalFlattenedTransposeDevice
            [d_kPointIndex * d_totalNonlocalElems * d_maxSingleAtomPseudoWfc *
             d_numberNodesPerElement])),
        d_maxSingleAtomPseudoWfc,
        strideB,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
        reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_cellHamMatrixTimesWaveMatrixNonLocalDevice[0])),
        numberWaveFunctions,
        strideC,
        d_totalNonlocalElems);


      for (unsigned int iAtom = 0; iAtom < d_totalNonlocalAtomsCurrentProc;
           ++iAtom)
        {
          const unsigned int accum = d_numberCellsAccumNonLocalAtoms[iAtom];
          addNonLocalContributionCUDAKernel<<<
            (numberWaveFunctions + 255) / 256 *
              d_numberCellsNonLocalAtoms[iAtom] * d_numberNodesPerElement,
            256>>>(
            numberWaveFunctions,
            d_numberCellsNonLocalAtoms[iAtom] * d_numberNodesPerElement,
            reinterpret_cast<const dataTypes::numberGPU *>(
              thrust::raw_pointer_cast(
                &d_cellHamMatrixTimesWaveMatrixNonLocalDevice[0])) +
              accum * d_numberNodesPerElement * numberWaveFunctions,
            reinterpret_cast<dataTypes::numberGPU *>(
              thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
            thrust::raw_pointer_cast(&d_cellNodeIdMapNonLocalToLocalDevice[0]) +
              accum * d_numberNodesPerElement);
        }
    }

  if (std::is_same<dataTypes::number, std::complex<double>>::value)
    {
      cudaUtils::copyComplexArrToRealArrsGPU(
        (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
         d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
        dst,
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]));


      daxpyAtomicAddKernel<<<(numberWaveFunctions + 255) / 256 *
                               d_numLocallyOwnedCells * d_numberNodesPerElement,
                             256>>>(
        numberWaveFunctions,
        d_numLocallyOwnedCells * d_numberNodesPerElement,
        reinterpret_cast<const dataTypes::numberGPU *>(
          thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]),
        thrust::raw_pointer_cast(
          &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));

      cudaUtils::copyRealArrsToComplexArrGPU(
        (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
         d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]),
        dst);
    }
  else
    daxpyAtomicAddKernel<<<(numberWaveFunctions + 255) / 256 *
                             d_numLocallyOwnedCells * d_numberNodesPerElement,
                           256>>>(
      numberWaveFunctions,
      d_numLocallyOwnedCells * d_numberNodesPerElement,
      reinterpret_cast<const dataTypes::numberGPU *>(
        thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
      dst,
      thrust::raw_pointer_cast(
        &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));
}


template <unsigned int FEOrder, unsigned int FEOrderElectro>
void
kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
  computeNonLocalProjectorKetTimesXTimesV(
    const dataTypes::numberGPU *             src,
    distributedGPUVec<dataTypes::numberGPU> &projectorKetTimesVector,
    const unsigned int                       numberWaveFunctions)
{
  const unsigned int totalLocallyOwnedCells =
    dftPtr->matrix_free_data.n_physical_cells();
  const dataTypes::number scalarCoeffAlpha = dataTypes::number(1.0),
                          scalarCoeffBeta  = dataTypes::number(0.0);

  //
  // compute C^{\dagger}*X
  //

  if (d_totalNonlocalElems > 0)
    {
      copyCUDAKernel<<<(numberWaveFunctions + 255) / 256 *
                         totalLocallyOwnedCells * d_numberNodesPerElement,
                       256>>>(
        numberWaveFunctions,
        totalLocallyOwnedCells * d_numberNodesPerElement,
        src,
        reinterpret_cast<dataTypes::numberGPU *>(
          thrust::raw_pointer_cast(&d_cellWaveFunctionMatrix[0])),
        thrust::raw_pointer_cast(
          &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));



      cublasXgemmBatched(
        d_cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numberWaveFunctions,
        d_maxSingleAtomPseudoWfc,
        d_numberNodesPerElement,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
        (const dataTypes::numberGPU **)d_A,
        numberWaveFunctions,
        (const dataTypes::numberGPU **)d_B,
        d_numberNodesPerElement,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
        d_C,
        numberWaveFunctions,
        d_totalNonlocalElems);

      cublasXgemm(
        d_cublasHandle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        numberWaveFunctions,
        d_totalPseudoWfcNonLocal,
        d_totalNonlocalElems * d_maxSingleAtomPseudoWfc,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
        reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsDevice[0])),
        numberWaveFunctions,
        reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorAllCellsReductionDevice[0])),
        d_totalNonlocalElems * d_maxSingleAtomPseudoWfc,
        reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
        reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
          &d_projectorKetTimesVectorParFlattenedDevice[0])),
        numberWaveFunctions);
    }

  projectorKetTimesVector.setZero();


  if (d_totalNonlocalElems > 0)
    copyToDealiiParallelNonLocalVec<<<(numberWaveFunctions + 255) / 256 *
                                        d_totalPseudoWfcNonLocal,
                                      256>>>(
      numberWaveFunctions,
      d_totalPseudoWfcNonLocal,
      reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
        &d_projectorKetTimesVectorParFlattenedDevice[0])),
      projectorKetTimesVector.begin(),
      thrust::raw_pointer_cast(&d_projectorIdsParallelNumberingMapDevice[0]));

  projectorKetTimesVector.compressAdd();
  projectorKetTimesVector.updateGhostValues();

  //
  // compute V*C^{\dagger}*X
  //
  if (d_totalNonlocalElems > 0)
    scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 *
                        d_totalPseudoWfcNonLocal,
                      256>>>(numberWaveFunctions,
                             d_totalPseudoWfcNonLocal,
                             1.0,
                             projectorKetTimesVector.begin(),
                             thrust::raw_pointer_cast(
                               &d_nonLocalPseudoPotentialConstantsDevice[0]));
}
