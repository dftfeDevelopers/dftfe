// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2022 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Phani Motamarri, Sambit Das
//


/** @file matrixVectorProductImplementations.cc
 *  @brief Contains linear algebra operations
 *
 */


template <unsigned int FEOrder, unsigned int FEOrderElectro>
void
kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
  computeLocalHamiltonianTimesX(const dataTypes::numberGPU *src,
                                const unsigned int          numberWaveFunctions,
                                dataTypes::numberGPU *      dst)
{
  const unsigned int kpointSpinIndex =
    (1 + dftParameters::spinPolarized) * d_kPointIndex + d_spinIndex;
  const unsigned int totalLocallyOwnedCells =
    dftPtr->matrix_free_data.n_physical_cells();

  copyCUDAKernel<<<(numberWaveFunctions + 255) / 256 * totalLocallyOwnedCells *
                     d_numberNodesPerElement,
                   256>>>(numberWaveFunctions,
                          totalLocallyOwnedCells * d_numberNodesPerElement,
                          src,
                          reinterpret_cast<dataTypes::numberGPU *>(
                            thrust::raw_pointer_cast(
                              &d_cellWaveFunctionMatrix[0])),
                          thrust::raw_pointer_cast(
                            &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));


  const dataTypes::number scalarCoeffAlpha = dataTypes::number(1.0),
                          scalarCoeffBeta  = dataTypes::number(0.0);
  const unsigned int strideA = d_numberNodesPerElement * numberWaveFunctions;
  const unsigned int strideB =
    d_numberNodesPerElement * d_numberNodesPerElement;
  const unsigned int strideC = d_numberNodesPerElement * numberWaveFunctions;


  cublasXgemmStridedBatched(
    d_cublasHandle,
    HIPBLAS_OP_N,
    std::is_same<dataTypes::number, std::complex<double>>::value ? HIPBLAS_OP_T :
                                                                   HIPBLAS_OP_N,
    numberWaveFunctions,
    d_numberNodesPerElement,
    d_numberNodesPerElement,
    reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffAlpha),
    reinterpret_cast<const dataTypes::numberGPU *>(
      thrust::raw_pointer_cast(&d_cellWaveFunctionMatrix[0])),
    numberWaveFunctions,
    strideA,
    reinterpret_cast<const dataTypes::numberGPU *>(thrust::raw_pointer_cast(
      &d_cellHamiltonianMatrixFlattenedDevice[d_numLocallyOwnedCells *
                                              d_numberNodesPerElement *
                                              d_numberNodesPerElement *
                                              kpointSpinIndex])),
    d_numberNodesPerElement,
    strideB,
    reinterpret_cast<const dataTypes::numberGPU *>(&scalarCoeffBeta),
    reinterpret_cast<dataTypes::numberGPU *>(
      thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
    numberWaveFunctions,
    strideC,
    totalLocallyOwnedCells);


  if (!(dftParameters::isPseudopotential &&
        dftPtr->d_nonLocalAtomGlobalChargeIds.size() > 0))
    {
      if (std::is_same<dataTypes::number, std::complex<double>>::value)
        {
          cudaUtils::copyComplexArrToRealArrsGPU(
            (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
             d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
            dst,
            thrust::raw_pointer_cast(&d_tempRealVec[0]),
            thrust::raw_pointer_cast(&d_tempImagVec[0]));


          daxpyAtomicAddKernel<<<(numberWaveFunctions + 255) / 256 *
                                   d_numLocallyOwnedCells *
                                   d_numberNodesPerElement,
                                 256>>>(
            numberWaveFunctions,
            d_numLocallyOwnedCells * d_numberNodesPerElement,
            reinterpret_cast<const dataTypes::numberGPU *>(
              thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
            thrust::raw_pointer_cast(&d_tempRealVec[0]),
            thrust::raw_pointer_cast(&d_tempImagVec[0]),
            thrust::raw_pointer_cast(
              &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));

          cudaUtils::copyRealArrsToComplexArrGPU(
            (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
             d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
            thrust::raw_pointer_cast(&d_tempRealVec[0]),
            thrust::raw_pointer_cast(&d_tempImagVec[0]),
            dst);
        }
      else
        daxpyAtomicAddKernel<<<(numberWaveFunctions + 255) / 256 *
                                 d_numLocallyOwnedCells *
                                 d_numberNodesPerElement,
                               256>>>(
          numberWaveFunctions,
          d_numLocallyOwnedCells * d_numberNodesPerElement,
          reinterpret_cast<const dataTypes::numberGPU *>(
            thrust::raw_pointer_cast(&d_cellHamMatrixTimesWaveMatrix[0])),
          dst,
          thrust::raw_pointer_cast(
            &d_flattenedArrayCellLocalProcIndexIdMapDevice[0]));
    }
}
