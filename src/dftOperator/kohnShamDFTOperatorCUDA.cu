#include "hip/hip_runtime.h"
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2022 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Phani Motamarri, Sambit Das
//

#include <cudaHelpers.h>
#include <kohnShamDFTOperatorCUDA.h>
#include <linearAlgebraOperations.h>
#include <linearAlgebraOperationsInternal.h>
#include <linearAlgebraOperationsCUDA.h>
#include <vectorUtilities.h>
#include <dft.h>
#include <dftParameters.h>
#include <dftUtils.h>


namespace dftfe
{
  namespace
  {
    /*
    #if __CUDA_ARCH__ < 600
        __device__ double
        atomicAdd(double *address, double val)
        {
          unsigned long long int *address_as_ull =
            (unsigned long long int *)address;
          unsigned long long int old = *address_as_ull, assumed;

          do
            {
              assumed = old;
              old     = atomicCAS(address_as_ull,
                              assumed,
                              __double_as_longlong(val +
                                                   __longlong_as_double(assumed)));

              // Note: uses integer comparison to avoid hang in case of NaN
    (since
              // NaN != NaN)
            }
          while (assumed != old);

          return __longlong_as_double(old);
        }
    #endif
    */


    __global__ void
    scaleCUDAKernel(const unsigned int contiguousBlockSize,
                    const unsigned int numContiguousBlocks,
                    const double       scalar,
                    double *           srcArray,
                    const double *     scalingVector)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerContiguousBlock =
        (contiguousBlockSize + (blockDim.x - 1)) / blockDim.x;
      const unsigned int gangBlockId = blockIdx.x / numGangsPerContiguousBlock;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerContiguousBlock * blockDim.x;
      if (globalThreadId <
            numContiguousBlocks * numGangsPerContiguousBlock * blockDim.x &&
          localThreadId < contiguousBlockSize)
        {
          *(srcArray + (localThreadId + gangBlockId * contiguousBlockSize)) =
            *(srcArray + (localThreadId + gangBlockId * contiguousBlockSize)) *
            (*(scalingVector + gangBlockId) * scalar);
        }
    }

    __global__ void
    scaleCUDAKernel(const unsigned int contiguousBlockSize,
                    const unsigned int numContiguousBlocks,
                    const double       scalar,
                    hipDoubleComplex *  srcArray,
                    const double *     scalingVector)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerContiguousBlock =
        (contiguousBlockSize + (blockDim.x - 1)) / blockDim.x;
      const unsigned int gangBlockId = blockIdx.x / numGangsPerContiguousBlock;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerContiguousBlock * blockDim.x;
      if (globalThreadId <
            numContiguousBlocks * numGangsPerContiguousBlock * blockDim.x &&
          localThreadId < contiguousBlockSize)
        {
          *(srcArray + (localThreadId + gangBlockId * contiguousBlockSize)) =
            hipCmul(
              *(srcArray + (localThreadId + gangBlockId * contiguousBlockSize)),
              make_hipDoubleComplex((*(scalingVector + gangBlockId) * scalar),
                                   0.0));
        }
    }

    template <typename numberType>
    __global__ void
    stridedCopyToBlockKernel(const unsigned int BVec,
                             const unsigned int M,
                             const numberType * xVec,
                             const unsigned int N,
                             numberType *       yVec,
                             const unsigned int startingXVecId)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerBVec = (BVec + blockDim.x - 1) / blockDim.x;
      const unsigned int gangBlockId     = blockIdx.x / numGangsPerBVec;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerBVec * blockDim.x;

      if (globalThreadId < M * numGangsPerBVec * blockDim.x &&
          localThreadId < BVec)
        {
          *(yVec + gangBlockId * BVec + localThreadId) =
            *(xVec + gangBlockId * N + startingXVecId + localThreadId);
        }
    }


    template <typename numberType>
    __global__ void
    stridedCopyFromBlockKernel(const unsigned int BVec,
                               const unsigned int M,
                               const numberType * xVec,
                               const unsigned int N,
                               numberType *       yVec,
                               const unsigned int startingXVecId)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerBVec = (BVec + blockDim.x - 1) / blockDim.x;
      const unsigned int gangBlockId     = blockIdx.x / numGangsPerBVec;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerBVec * blockDim.x;

      if (globalThreadId < M * numGangsPerBVec * blockDim.x &&
          localThreadId < BVec)
        {
          *(yVec + gangBlockId * N + startingXVecId + localThreadId) =
            *(xVec + gangBlockId * BVec + localThreadId);
        }
    }

    __global__ void
    stridedCopyFromBlockKernelFP32(const unsigned int BVec,
                                   const unsigned int M,
                                   const double *     xVec,
                                   const unsigned int N,
                                   float *            yVec,
                                   const unsigned int startingXVecId)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerBVec = (BVec + blockDim.x - 1) / blockDim.x;
      const unsigned int gangBlockId     = blockIdx.x / numGangsPerBVec;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerBVec * blockDim.x;

      if (globalThreadId < M * numGangsPerBVec * blockDim.x &&
          localThreadId < BVec)
        {
          *(yVec + gangBlockId * N + startingXVecId + localThreadId) =
            *(xVec + gangBlockId * BVec + localThreadId);
        }
    }

    __global__ void
    stridedCopyFromBlockKernelFP32(const unsigned int     BVec,
                                   const unsigned int     M,
                                   const hipDoubleComplex *xVec,
                                   const unsigned int     N,
                                   hipFloatComplex *       yVec,
                                   const unsigned int     startingXVecId)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numGangsPerBVec = (BVec + blockDim.x - 1) / blockDim.x;
      const unsigned int gangBlockId     = blockIdx.x / numGangsPerBVec;
      const unsigned int localThreadId =
        globalThreadId - gangBlockId * numGangsPerBVec * blockDim.x;

      if (globalThreadId < M * numGangsPerBVec * blockDim.x &&
          localThreadId < BVec)
        {
          *(yVec + gangBlockId * N + startingXVecId + localThreadId) =
            hipComplexDoubleToFloat(
              *(xVec + gangBlockId * BVec + localThreadId));
        }
    }


    __global__ void
    convDoubleArrToFloatArr(const unsigned int size,
                            const double *     doubleArr,
                            float *            floatArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

      for (unsigned int index = globalThreadId; index < size;
           index += blockDim.x * gridDim.x)
        floatArr[index] = doubleArr[index];
    }

    __global__ void
    convDoubleArrToFloatArr(const unsigned int     size,
                            const hipDoubleComplex *doubleArr,
                            hipFloatComplex *       floatArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

      for (unsigned int index = globalThreadId; index < size;
           index += blockDim.x * gridDim.x)
        floatArr[index] = hipComplexDoubleToFloat(doubleArr[index]);
    }


    __global__ void
    convFloatArrToDoubleArr(const unsigned int size,
                            const float *      floatArr,
                            double *           doubleArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

      for (unsigned int index = globalThreadId; index < size;
           index += blockDim.x * gridDim.x)
        doubleArr[index] = floatArr[index];
    }


    __global__ void
    convFloatArrToDoubleArr(const unsigned int    size,
                            const hipFloatComplex *floatArr,
                            hipDoubleComplex *     doubleArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

      for (unsigned int index = globalThreadId; index < size;
           index += blockDim.x * gridDim.x)
        doubleArr[index] = hipComplexFloatToDouble(floatArr[index]);
    }


    __global__ void
    copyFloatArrToDoubleArrLocallyOwned(const unsigned int  contiguousBlockSize,
                                        const unsigned int  numContiguousBlocks,
                                        const float *       floatArr,
                                        const unsigned int *locallyOwnedFlagArr,
                                        double *            doubleArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex = index / contiguousBlockSize;
          if (locallyOwnedFlagArr[blockIndex] == 1)
            doubleArr[index] = floatArr[index];
        }
    }

    __global__ void
    copyFloatArrToDoubleArrLocallyOwned(const unsigned int contiguousBlockSize,
                                        const unsigned int numContiguousBlocks,
                                        const hipFloatComplex *floatArr,
                                        const unsigned int *locallyOwnedFlagArr,
                                        hipDoubleComplex *   doubleArr)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex = index / contiguousBlockSize;
          if (locallyOwnedFlagArr[blockIndex] == 1)
            doubleArr[index] = hipComplexFloatToDouble(floatArr[index]);
        }
    }

    template <typename numberType>
    __global__ void
    copyCUDAKernel(const unsigned int contiguousBlockSize,
                   const unsigned int numContiguousBlocks,
                   const numberType * copyFromVec,
                   numberType *       copyToVec,
                   const dealii::types::global_dof_index
                     *copyFromVecStartingContiguousBlockIds)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex      = index / contiguousBlockSize;
          unsigned int intraBlockIndex = index % contiguousBlockSize;
          copyToVec[index] =
            copyFromVec[copyFromVecStartingContiguousBlockIds[blockIndex] +
                        intraBlockIndex];
        }
    }

    __global__ void
    daxpyAtomicAddKernel(
      const unsigned int                     contiguousBlockSize,
      const unsigned int                     numContiguousBlocks,
      const double *                         addFromVec,
      double *                               addToVec,
      const dealii::types::global_dof_index *addToVecStartingContiguousBlockIds)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex      = index / contiguousBlockSize;
          unsigned int intraBlockIndex = index % contiguousBlockSize;
          atomicAdd(&addToVec[addToVecStartingContiguousBlockIds[blockIndex] +
                              intraBlockIndex],
                    addFromVec[index]);
        }
    }


    __global__ void
    daxpyAtomicAddKernel(
      const unsigned int                     contiguousBlockSize,
      const unsigned int                     numContiguousBlocks,
      const hipDoubleComplex *                addFromVec,
      hipDoubleComplex *                      addToVec,
      const dealii::types::global_dof_index *addToVecStartingContiguousBlockIds)
    {}


    __global__ void
    daxpyAtomicAddKernel(
      const unsigned int                     contiguousBlockSize,
      const unsigned int                     numContiguousBlocks,
      const double *                         addFromVec,
      double *                               addToVecReal,
      double *                               addToVecImag,
      const dealii::types::global_dof_index *addToVecStartingContiguousBlockIds)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex      = index / contiguousBlockSize;
          unsigned int intraBlockIndex = index % contiguousBlockSize;
          atomicAdd(
            &addToVecReal[addToVecStartingContiguousBlockIds[blockIndex] +
                          intraBlockIndex],
            addFromVec[index]);
          atomicAdd(
            &addToVecImag[addToVecStartingContiguousBlockIds[blockIndex] +
                          intraBlockIndex],
            addFromVec[index]);
        }
    }

    __global__ void
    daxpyAtomicAddKernel(
      const unsigned int                     contiguousBlockSize,
      const unsigned int                     numContiguousBlocks,
      const hipDoubleComplex *                addFromVec,
      double *                               addToVecReal,
      double *                               addToVecImag,
      const dealii::types::global_dof_index *addToVecStartingContiguousBlockIds)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          unsigned int blockIndex      = index / contiguousBlockSize;
          unsigned int intraBlockIndex = index % contiguousBlockSize;
          atomicAdd(
            &addToVecReal[addToVecStartingContiguousBlockIds[blockIndex] +
                          intraBlockIndex],
            addFromVec[index].x);
          atomicAdd(
            &addToVecImag[addToVecStartingContiguousBlockIds[blockIndex] +
                          intraBlockIndex],
            addFromVec[index].y);
        }
    }


    template <typename numberType>
    __global__ void
    copyToParallelNonLocalVecFromReducedVec(
      const unsigned int  numWfcs,
      const unsigned int  totalPseudoWfcs,
      const numberType *  reducedProjectorKetTimesWfcVec,
      numberType *        projectorKetTimesWfcParallelVec,
      const unsigned int *indexMapFromParallelVecToReducedVec)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries  = totalPseudoWfcs * numWfcs;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          const unsigned int blockIndex      = index / numWfcs;
          const unsigned int intraBlockIndex = index % numWfcs;
          // projectorKetTimesWfcParallelVec[index]
          //        =reducedProjectorKetTimesWfcVec[indexMapFromParallelVecToReducedVec[blockIndex]*numWfcs+intraBlockIndex];
          projectorKetTimesWfcParallelVec
            [indexMapFromParallelVecToReducedVec[blockIndex] * numWfcs +
             intraBlockIndex] = reducedProjectorKetTimesWfcVec[index];
        }
    }

    template <typename numberType>
    __global__ void
    copyFromParallelNonLocalVecToAllCellsVec(
      const unsigned int numWfcs,
      const unsigned int numNonLocalCells,
      const unsigned int maxSingleAtomPseudoWfc,
      const numberType * projectorKetTimesWfcParallelVec,
      numberType *       projectorKetTimesWfcAllCellsVec,
      const int *        indexMapPaddedToParallelVec)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries =
        numNonLocalCells * maxSingleAtomPseudoWfc * numWfcs;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          const unsigned int blockIndex      = index / numWfcs;
          const unsigned int intraBlockIndex = index % numWfcs;
          const int mappedIndex = indexMapPaddedToParallelVec[blockIndex];
          if (mappedIndex != -1)
            projectorKetTimesWfcAllCellsVec[index] =
              projectorKetTimesWfcParallelVec[mappedIndex * numWfcs +
                                              intraBlockIndex];
        }
    }


    template <typename numberType>
    __global__ void
    copyToDealiiParallelNonLocalVec(
      const unsigned int  numWfcs,
      const unsigned int  totalPseudoWfcs,
      const numberType *  projectorKetTimesWfcParallelVec,
      numberType *        projectorKetTimesWfcDealiiParallelVec,
      const unsigned int *indexMapDealiiParallelNumbering)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries  = totalPseudoWfcs * numWfcs;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          const unsigned int blockIndex      = index / numWfcs;
          const unsigned int intraBlockIndex = index % numWfcs;
          const unsigned int mappedIndex =
            indexMapDealiiParallelNumbering[blockIndex];

          projectorKetTimesWfcDealiiParallelVec[mappedIndex * numWfcs +
                                                intraBlockIndex] =
            projectorKetTimesWfcParallelVec[index];
        }
    }

    template <typename numberType>
    __global__ void
    copyFromDealiiParallelNonLocalVec(
      const unsigned int  numWfcs,
      const unsigned int  totalPseudoWfcs,
      numberType *        projectorKetTimesWfcParallelVec,
      const numberType *  projectorKetTimesWfcDealiiParallelVec,
      const unsigned int *indexMapDealiiParallelNumbering)
    {
      const unsigned int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
      const unsigned int numberEntries  = totalPseudoWfcs * numWfcs;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          const unsigned int blockIndex      = index / numWfcs;
          const unsigned int intraBlockIndex = index % numWfcs;
          const unsigned int mappedIndex =
            indexMapDealiiParallelNumbering[blockIndex];

          projectorKetTimesWfcParallelVec[index] =
            projectorKetTimesWfcDealiiParallelVec[mappedIndex * numWfcs +
                                                  intraBlockIndex];
        }
    }

    __global__ void
    addNonLocalContributionCUDAKernel(
      const dealii::types::global_dof_index contiguousBlockSize,
      const dealii::types::global_dof_index numContiguousBlocks,
      const double *                        xVec,
      double *                              yVec,
      const unsigned int *                  xVecToyVecBlockIdMap)
    {
      const dealii::types::global_dof_index globalThreadId =
        blockIdx.x * blockDim.x + threadIdx.x;
      const dealii::types::global_dof_index numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          dealii::types::global_dof_index blockIndex =
            index / contiguousBlockSize;
          dealii::types::global_dof_index intraBlockIndex =
            index % contiguousBlockSize;
          yVec[xVecToyVecBlockIdMap[blockIndex] * contiguousBlockSize +
               intraBlockIndex] += xVec[index];
        }
    }

    __global__ void
    addNonLocalContributionCUDAKernel(
      const dealii::types::global_dof_index contiguousBlockSize,
      const dealii::types::global_dof_index numContiguousBlocks,
      const hipDoubleComplex *               xVec,
      hipDoubleComplex *                     yVec,
      const unsigned int *                  xVecToyVecBlockIdMap)
    {
      const dealii::types::global_dof_index globalThreadId =
        blockIdx.x * blockDim.x + threadIdx.x;
      const dealii::types::global_dof_index numberEntries =
        numContiguousBlocks * contiguousBlockSize;

      for (unsigned int index = globalThreadId; index < numberEntries;
           index += blockDim.x * gridDim.x)
        {
          dealii::types::global_dof_index blockIndex =
            index / contiguousBlockSize;
          dealii::types::global_dof_index intraBlockIndex =
            index % contiguousBlockSize;
          yVec[xVecToyVecBlockIdMap[blockIndex] * contiguousBlockSize +
               intraBlockIndex] =
            hipCadd(yVec[xVecToyVecBlockIdMap[blockIndex] * contiguousBlockSize +
                        intraBlockIndex],
                   xVec[index]);
        }
    }
  } // namespace

  //
  // constructor
  //
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    kohnShamDFTOperatorCUDAClass(dftClass<FEOrder, FEOrderElectro> *_dftPtr,
                                 const MPI_Comm &mpi_comm_replica)
    : dftPtr(_dftPtr)
    , d_kPointIndex(0)
    , d_numberNodesPerElement(_dftPtr->matrix_free_data.get_dofs_per_cell())
    , d_numberMacroCells(_dftPtr->matrix_free_data.n_macro_cells())
    , d_numLocallyOwnedCells(dftPtr->matrix_free_data.n_physical_cells())
    , d_numQuadPoints(
        dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId)
          .size())
    , d_isStiffnessMatrixExternalPotCorrComputed(false)
    , d_isMallocCalled(false)
    , mpi_communicator(mpi_comm_replica)
    , n_mpi_processes(Utilities::MPI::n_mpi_processes(mpi_comm_replica))
    , this_mpi_process(Utilities::MPI::this_mpi_process(mpi_comm_replica))
    , pcout(std::cout, (Utilities::MPI::this_mpi_process(MPI_COMM_WORLD) == 0))
    , computing_timer(mpi_comm_replica,
                      pcout,
                      TimerOutput::never,
                      TimerOutput::wall_times)
    , operatorDFTCUDAClass(mpi_comm_replica,
                           _dftPtr->getMatrixFreeData(),
                           _dftPtr->constraintsNoneDataInfo,
                           _dftPtr->d_constraintsNoneDataInfoCUDA)
  {}

  //
  // destructor
  //
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::~kohnShamDFTOperatorCUDAClass()
  {
    if (d_isMallocCalled)
      {
        free(h_d_A);
        free(h_d_B);
        free(h_d_C);
        CUDACHECK(hipFree(d_A));
        CUDACHECK(hipFree(d_B));
        CUDACHECK(hipFree(d_C));
      }
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::createCublasHandle()
  {
    hipblasCreate(&d_cublasHandle);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::destroyCublasHandle()
  {
    hipblasDestroy(d_cublasHandle);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  hipblasHandle_t &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::getCublasHandle()
  {
    return d_cublasHandle;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  const double *
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::getSqrtMassVec()
  {
    return thrust::raw_pointer_cast(&d_sqrtMassVectorDevice[0]);
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  const double *
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::getInvSqrtMassVec()
  {
    return thrust::raw_pointer_cast(&d_invSqrtMassVectorDevice[0]);
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedCPUVec<dataTypes::number> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getProjectorKetTimesVectorSingle()
  {
    return dftPtr->d_projectorKetTimesVectorPar[0];
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientIntegral()
  {
    return d_cellShapeFunctionGradientIntegralFlattenedDevice;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientIntegralElectro()
  {
    return d_cellShapeFunctionGradientIntegralFlattenedDeviceElectro;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::getShapeFunctionValues()
  {
    return d_shapeFunctionValueDevice;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionValuesInverted(const bool use2pPlusOneGLQuad)
  {
    return use2pPlusOneGLQuad ? d_glShapeFunctionValueInvertedDevice :
                                d_shapeFunctionValueInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionValuesNLPInverted()
  {
    return d_shapeFunctionValueNLPInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientValuesXInverted()
  {
    return d_shapeFunctionGradientValueXInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientValuesYInverted()
  {
    return d_shapeFunctionGradientValueYInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientValuesZInverted()
  {
    return d_shapeFunctionGradientValueZInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getShapeFunctionGradientValuesNLPInverted()
  {
    return d_shapeFunctionGradientValueNLPInvertedDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<double> &
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::getInverseJacobiansNLP()
  {
    return d_inverseJacobiansNLPDevice;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<dealii::types::global_dof_index> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getFlattenedArrayCellLocalProcIndexIdMap()
  {
    return d_flattenedArrayCellLocalProcIndexIdMapDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<dataTypes::numberThrustGPU> &
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::getCellWaveFunctionMatrix()
  {
    return d_cellWaveFunctionMatrix;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedCPUVec<dataTypes::number> &
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::getParallelVecSingleComponent()
  {
    return d_parallelVecSingleComponent;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedGPUVec<dataTypes::numberGPU> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getParallelChebyBlockVectorDevice()
  {
    return d_parallelChebyBlockVectorDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  distributedGPUVec<dataTypes::numberGPU> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getParallelProjectorKetTimesBlockVectorDevice()
  {
    return d_parallelProjectorKetTimesBlockVectorDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<unsigned int> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getLocallyOwnedProcBoundaryNodesVectorDevice()
  {
    return d_locallyOwnedProcBoundaryNodesVectorDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<unsigned int> &
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    getLocallyOwnedProcProjectorKetBoundaryNodesVectorDevice()
  {
    return d_locallyOwnedProcProjectorKetBoundaryNodesVectorDevice;
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  thrust::device_vector<unsigned int> &
  kohnShamDFTOperatorCUDAClass<FEOrder,
                               FEOrderElectro>::getBoundaryIdToLocalIdMap()
  {
    return d_boundaryIdToLocalIdMapDevice;
  }

  //
  // initialize kohnShamDFTOperatorCUDAClass object
  //
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::init()
  {
    computing_timer.enter_subsection("kohnShamDFTOperatorCUDAClass setup");


    dftPtr->matrix_free_data.initialize_dof_vector(
      d_invSqrtMassVector, dftPtr->d_densityDofHandlerIndex);
    d_sqrtMassVector.reinit(d_invSqrtMassVector);



    //
    // compute mass vector
    //
    computeMassVector(dftPtr->dofHandler,
                      dftPtr->constraintsNone,
                      d_sqrtMassVector,
                      d_invSqrtMassVector);

    computing_timer.leave_subsection("kohnShamDFTOperatorCUDAClass setup");
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::resetExtPotHamFlag()
  {
    d_isStiffnessMatrixExternalPotCorrComputed = false;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::reinit(
    const unsigned int numberWaveFunctions,
    bool               flag)
  {
    distributedCPUVec<dataTypes::number> flattenedArray;
    if (flag)
      vectorTools::createDealiiVector<dataTypes::number>(
        dftPtr->matrix_free_data.get_vector_partitioner(
          dftPtr->d_densityDofHandlerIndex),
        numberWaveFunctions,
        flattenedArray);

    vectorTools::createDealiiVector<dataTypes::number>(
      dftPtr->matrix_free_data.get_vector_partitioner(
        dftPtr->d_densityDofHandlerIndex),
      1,
      d_parallelVecSingleComponent);

    size_t free_t, total_t;

    hipMemGetInfo(&free_t, &total_t);
    if (dftParameters::verbosity >= 2)
      pcout << "starting free mem: " << free_t << ", total mem: " << total_t
            << std::endl;

    const unsigned int BVec =
      std::min(dftParameters::chebyWfcBlockSize, numberWaveFunctions);
    d_parallelChebyBlockVectorDevice.reinit(
      dftPtr->matrix_free_data.get_vector_partitioner(
        dftPtr->d_densityDofHandlerIndex),
      BVec);

    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      {
        d_tempRealVec.resize(
          (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
           d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
          0.0);
        d_tempImagVec.resize(
          (d_parallelChebyBlockVectorDevice.locallyOwnedFlattenedSize() +
           d_parallelChebyBlockVectorDevice.ghostFlattenedSize()),
          0.0);
      }

    const unsigned int n_ghosts =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->n_ghost_indices();
    const unsigned int localSize =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->local_size();

    thrust::host_vector<unsigned int> locallyOwnedProcBoundaryNodesVector(
      localSize, 0);

    const std::vector<std::pair<unsigned int, unsigned int>>
      &locallyOwnedProcBoundaryNodes =
        dftPtr->matrix_free_data
          .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
          ->import_indices();

    for (unsigned int iset = 0; iset < locallyOwnedProcBoundaryNodes.size();
         ++iset)
      {
        const std::pair<unsigned int, unsigned int> &localIndices =
          locallyOwnedProcBoundaryNodes[iset];
        for (unsigned int inode = localIndices.first;
             inode < localIndices.second;
             ++inode)
          {
            locallyOwnedProcBoundaryNodesVector[inode] = 1;
          }
      }

    d_locallyOwnedProcBoundaryNodesVectorDevice.resize(localSize);

    /*hipMemcpy(thrust::raw_pointer_cast(&d_locallyOwnedProcBoundaryNodesVectorDevice[0]),
      locallyOwnedProcBoundaryNodesVector.begin(),
      localSize*sizeof(unsigned int),
      hipMemcpyHostToDevice);*/

    d_locallyOwnedProcBoundaryNodesVectorDevice =
      locallyOwnedProcBoundaryNodesVector;


    vectorTools::computeCellLocalIndexSetMap(
      flattenedArray.get_partitioner(),
      dftPtr->matrix_free_data,
      dftPtr->d_densityDofHandlerIndex,
      numberWaveFunctions,
      d_flattenedArrayMacroCellLocalProcIndexIdMapFlattened,
      d_normalCellIdToMacroCellIdMap,
      d_macroCellIdToNormalCellIdMap,
      d_flattenedArrayCellLocalProcIndexIdMap);

    d_flattenedArrayCellLocalProcIndexIdMapDevice =
      d_flattenedArrayCellLocalProcIndexIdMap;



    getOverloadedConstraintMatrix()->precomputeMaps(
      dftPtr->matrix_free_data.get_vector_partitioner(
        dftPtr->d_densityDofHandlerIndex),
      flattenedArray.get_partitioner(),
      numberWaveFunctions);

    getOverloadedConstraintMatrixHost()->precomputeMaps(
      dftPtr->matrix_free_data.get_vector_partitioner(),
      dftPtr->matrix_free_data.get_vector_partitioner(),
      1);


    const unsigned int totalLocallyOwnedCells =
      dftPtr->matrix_free_data.n_physical_cells();

    d_cellHamiltonianMatrixFlattenedDevice.resize(
      d_numLocallyOwnedCells * d_numberNodesPerElement *
        d_numberNodesPerElement * dftPtr->d_kPointWeights.size() *
        (1 + dftParameters::spinPolarized),
      dataTypes::numberThrustGPU(0.0));

    if (dftParameters::isPseudopotential)
      d_cellHamiltonianMatrixExternalPotCorrFlattenedDevice.resize(
        d_numLocallyOwnedCells * d_numberNodesPerElement *
          d_numberNodesPerElement,
        0.0);
    else
      d_cellHamiltonianMatrixExternalPotCorrFlattenedDevice.resize(10, 0.0);

    d_cellWaveFunctionMatrix.resize(totalLocallyOwnedCells *
                                      d_numberNodesPerElement *
                                      numberWaveFunctions,
                                    0.0);

    d_cellHamMatrixTimesWaveMatrix.resize(totalLocallyOwnedCells *
                                            d_numberNodesPerElement *
                                            numberWaveFunctions,
                                          0.0);

    if (dftParameters::isPseudopotential)
      {
        d_parallelProjectorKetTimesBlockVectorDevice.reinit(
          dftPtr->d_projectorKetTimesVectorPar[0].get_partitioner(), BVec);


        thrust::host_vector<unsigned int>
          locallyOwnedProcProjectorKetBoundaryNodesVector(
            dftPtr->d_projectorKetTimesVectorPar[0]
              .get_partitioner()
              ->local_size(),
            0);

        const std::vector<std::pair<unsigned int, unsigned int>>
          &locallyOwnedProcProjectorKetBoundaryNodes =
            dftPtr->d_projectorKetTimesVectorPar[0]
              .get_partitioner()
              ->import_indices();

        for (unsigned int iset = 0;
             iset < locallyOwnedProcProjectorKetBoundaryNodes.size();
             ++iset)
          {
            const std::pair<unsigned int, unsigned int> &localIndices =
              locallyOwnedProcProjectorKetBoundaryNodes[iset];
            for (unsigned int inode = localIndices.first;
                 inode < localIndices.second;
                 ++inode)
              {
                locallyOwnedProcProjectorKetBoundaryNodesVector[inode] = 1;
              }
          }

        d_locallyOwnedProcProjectorKetBoundaryNodesVectorDevice.resize(
          dftPtr->d_projectorKetTimesVectorPar[0]
            .get_partitioner()
            ->local_size());
        d_locallyOwnedProcProjectorKetBoundaryNodesVectorDevice =
          locallyOwnedProcProjectorKetBoundaryNodesVector;


        d_totalPseudoWfcNonLocal = 0;
        d_totalNonlocalElems     = 0;
        d_totalNonlocalAtomsCurrentProc =
          dftPtr->d_nonLocalAtomIdsInCurrentProcess.size();
        unsigned int maxPseudoWfc = 0;
        d_numberCellsAccumNonLocalAtoms.resize(d_totalNonlocalAtomsCurrentProc);
        std::vector<unsigned int> numPseduoWfcsAccum(
          d_totalNonlocalAtomsCurrentProc);
        for (unsigned int iAtom = 0;
             iAtom < dftPtr->d_nonLocalAtomIdsInCurrentProcess.size();
             ++iAtom)
          {
            const unsigned int atomId =
              dftPtr->d_nonLocalAtomIdsInCurrentProcess[iAtom];
            const unsigned int numberSingleAtomPseudoWaveFunctions =
              dftPtr->d_numberPseudoAtomicWaveFunctions[atomId];
            if (numberSingleAtomPseudoWaveFunctions > maxPseudoWfc)
              maxPseudoWfc = numberSingleAtomPseudoWaveFunctions;

            numPseduoWfcsAccum[iAtom] = d_totalPseudoWfcNonLocal;
            d_totalPseudoWfcNonLocal += numberSingleAtomPseudoWaveFunctions;
            const unsigned int numberElementsInCompactSupport =
              dftPtr->d_elementIteratorsInAtomCompactSupport[atomId].size();
            d_numberCellsAccumNonLocalAtoms[iAtom] = d_totalNonlocalElems;
            d_totalNonlocalElems += numberElementsInCompactSupport;
          }

        d_maxSingleAtomPseudoWfc = maxPseudoWfc;
        d_cellHamMatrixTimesWaveMatrixNonLocalDevice.resize(
          d_totalNonlocalElems * numberWaveFunctions * d_numberNodesPerElement,
          dataTypes::numberThrustGPU(0.0));
        d_cellHamiltonianMatrixNonLocalFlattenedConjugate.clear();
        d_cellHamiltonianMatrixNonLocalFlattenedConjugate.resize(
          dftPtr->d_kPointWeights.size() * d_totalNonlocalElems *
            d_numberNodesPerElement * d_maxSingleAtomPseudoWfc,
          dataTypes::number(0.0));
        d_cellHamiltonianMatrixNonLocalFlattenedTranspose.clear();
        d_cellHamiltonianMatrixNonLocalFlattenedTranspose.resize(
          dftPtr->d_kPointWeights.size() * d_totalNonlocalElems *
            d_numberNodesPerElement * d_maxSingleAtomPseudoWfc,
          dataTypes::number(0.0));
        d_nonLocalPseudoPotentialConstants.clear();
        d_nonLocalPseudoPotentialConstants.resize(d_totalPseudoWfcNonLocal,
                                                  0.0);
        d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocal.clear();
        d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocal.resize(
          d_totalNonlocalElems * d_numberNodesPerElement, 0);
        d_projectorKetTimesVectorAllCellsDevice.resize(
          d_totalNonlocalElems * numberWaveFunctions * d_maxSingleAtomPseudoWfc,
          dataTypes::numberThrustGPU(0.0));

        d_projectorIdsParallelNumberingMap.clear();
        d_projectorIdsParallelNumberingMap.resize(d_totalPseudoWfcNonLocal, 0);
        d_projectorKetTimesVectorParFlattenedDevice.resize(
          numberWaveFunctions * d_totalPseudoWfcNonLocal, 0.0);

        d_indexMapFromPaddedNonLocalVecToParallelNonLocalVec.clear();
        d_indexMapFromPaddedNonLocalVecToParallelNonLocalVec.resize(
          d_totalNonlocalElems * d_maxSingleAtomPseudoWfc, -1);

        d_nonlocalElemIdToLocalElemIdMap.clear();
        d_nonlocalElemIdToLocalElemIdMap.resize(d_totalNonlocalElems, 0);

        d_projectorKetTimesVectorAllCellsReduction.clear();
        d_projectorKetTimesVectorAllCellsReduction.resize(
          d_totalNonlocalElems * d_maxSingleAtomPseudoWfc *
            d_totalPseudoWfcNonLocal,
          dataTypes::number(0.0));

        d_cellNodeIdMapNonLocalToLocal.clear();
        d_cellNodeIdMapNonLocalToLocal.resize(d_totalNonlocalElems *
                                              d_numberNodesPerElement);

        unsigned int countElemNode   = 0;
        unsigned int countElem       = 0;
        unsigned int countPseudoWfc1 = 0;
        d_numberCellsNonLocalAtoms.resize(d_totalNonlocalAtomsCurrentProc);
        for (unsigned int iAtom = 0; iAtom < d_totalNonlocalAtomsCurrentProc;
             ++iAtom)
          {
            const unsigned int atomId =
              dftPtr->d_nonLocalAtomIdsInCurrentProcess[iAtom];
            const unsigned int numberPseudoWaveFunctions =
              dftPtr->d_numberPseudoAtomicWaveFunctions[atomId];

            d_numberCellsNonLocalAtoms[iAtom] =
              dftPtr->d_elementIteratorsInAtomCompactSupport[atomId].size();

            for (unsigned int ipseudowfc = 0;
                 ipseudowfc < numberPseudoWaveFunctions;
                 ++ipseudowfc)
              {
                const unsigned int id =
                  dftPtr->d_projectorKetTimesVectorPar[0]
                    .get_partitioner()
                    ->global_to_local(
                      dftPtr->d_projectorIdsNumberingMapCurrentProcess
                        [std::make_pair(atomId, ipseudowfc)]);

                d_projectorIdsParallelNumberingMap[countPseudoWfc1] = id;
                // std::cout<<"iAtom: "<< iAtom<<", ipseudo: "<< ipseudowfc <<",
                // netpseudo: "<<countPseudoWfc1<<", parallel id:
                // "<<id<<std::endl;
                // d_nonLocalPseudoPotentialConstants[countPseudoWfc1]
                //   =dftPtr->d_nonLocalPseudoPotentialConstants[atomId][ipseudowfc];
                d_nonLocalPseudoPotentialConstants[id] =
                  dftPtr
                    ->d_nonLocalPseudoPotentialConstants[atomId][ipseudowfc];
                for (unsigned int iElemComp = 0;
                     iElemComp <
                     dftPtr->d_elementIteratorsInAtomCompactSupport[atomId]
                       .size();
                     ++iElemComp)
                  d_indexMapFromPaddedNonLocalVecToParallelNonLocalVec
                    [d_numberCellsAccumNonLocalAtoms[iAtom] *
                       d_maxSingleAtomPseudoWfc +
                     iElemComp * d_maxSingleAtomPseudoWfc + ipseudowfc] =
                      id; // countPseudoWfc1;//id;

                countPseudoWfc1++;
              }

            for (unsigned int iElemComp = 0;
                 iElemComp <
                 dftPtr->d_elementIteratorsInAtomCompactSupport[atomId].size();
                 ++iElemComp)
              {
                const unsigned int elementId =
                  dftPtr->d_elementIdsInAtomCompactSupport[atomId][iElemComp];
                for (unsigned int iNode = 0; iNode < d_numberNodesPerElement;
                     ++iNode)
                  {
                    dealii::types::global_dof_index localNodeId =
                      d_flattenedArrayCellLocalProcIndexIdMap
                        [elementId * d_numberNodesPerElement + iNode];
                    d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocal
                      [countElemNode] = localNodeId;
                    d_cellNodeIdMapNonLocalToLocal[countElemNode] =
                      elementId * d_numberNodesPerElement + iNode;
                    countElemNode++;
                  }
              }

            for (unsigned int iElemComp = 0;
                 iElemComp <
                 dftPtr->d_elementIteratorsInAtomCompactSupport[atomId].size();
                 ++iElemComp)
              {
                const unsigned int elementId =
                  dftPtr->d_elementIdsInAtomCompactSupport[atomId][iElemComp];
                d_nonlocalElemIdToLocalElemIdMap[countElem] = elementId;

                for (unsigned int ikpoint = 0;
                     ikpoint < dftPtr->d_kPointWeights.size();
                     ikpoint++)
                  for (unsigned int iNode = 0; iNode < d_numberNodesPerElement;
                       ++iNode)
                    {
                      for (unsigned int iPseudoWave = 0;
                           iPseudoWave < numberPseudoWaveFunctions;
                           ++iPseudoWave)
                        {
                          d_cellHamiltonianMatrixNonLocalFlattenedConjugate
                            [ikpoint * d_totalNonlocalElems *
                               d_numberNodesPerElement *
                               d_maxSingleAtomPseudoWfc +
                             countElem * d_maxSingleAtomPseudoWfc *
                               d_numberNodesPerElement +
                             d_numberNodesPerElement * iPseudoWave + iNode] =
                              dftPtr
                                ->d_nonLocalProjectorElementMatricesConjugate
                                  [atomId][iElemComp]
                                  [ikpoint * d_numberNodesPerElement *
                                     numberPseudoWaveFunctions +
                                   d_numberNodesPerElement * iPseudoWave +
                                   iNode];

                          d_cellHamiltonianMatrixNonLocalFlattenedTranspose
                            [ikpoint * d_totalNonlocalElems *
                               d_numberNodesPerElement *
                               d_maxSingleAtomPseudoWfc +
                             countElem * d_numberNodesPerElement *
                               d_maxSingleAtomPseudoWfc +
                             d_maxSingleAtomPseudoWfc * iNode + iPseudoWave] =
                              dftPtr
                                ->d_nonLocalProjectorElementMatricesTranspose
                                  [atomId][iElemComp]
                                  [ikpoint * d_numberNodesPerElement *
                                     numberPseudoWaveFunctions +
                                   numberPseudoWaveFunctions * iNode +
                                   iPseudoWave];
                        }
                    }


                for (unsigned int iPseudoWave = 0;
                     iPseudoWave < numberPseudoWaveFunctions;
                     ++iPseudoWave)
                  {
                    const unsigned int columnStartId =
                      (numPseduoWfcsAccum[iAtom] + iPseudoWave) *
                      d_totalNonlocalElems * d_maxSingleAtomPseudoWfc;
                    const unsigned int columnRowId =
                      countElem * d_maxSingleAtomPseudoWfc + iPseudoWave;
                    d_projectorKetTimesVectorAllCellsReduction[columnStartId +
                                                               columnRowId] =
                      dataTypes::number(1.0);
                  }

                countElem++;
              }
          }

        d_cellHamiltonianMatrixNonLocalFlattenedConjugateDevice =
          d_cellHamiltonianMatrixNonLocalFlattenedConjugate;
        d_cellHamiltonianMatrixNonLocalFlattenedTransposeDevice =
          d_cellHamiltonianMatrixNonLocalFlattenedTranspose;
        d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocalDevice =
          d_flattenedArrayCellLocalProcIndexIdFlattenedMapNonLocal;
        d_projectorIdsParallelNumberingMapDevice =
          d_projectorIdsParallelNumberingMap;
        // d_indexMapFromParallelNonLocalVecToReducedVecDevice=d_indexMapFromParallelNonLocalVecToReducedVec;
        d_indexMapFromPaddedNonLocalVecToParallelNonLocalVecDevice =
          d_indexMapFromPaddedNonLocalVecToParallelNonLocalVec;
        d_projectorKetTimesVectorAllCellsReductionDevice =
          d_projectorKetTimesVectorAllCellsReduction;
        d_nonLocalPseudoPotentialConstantsDevice =
          d_nonLocalPseudoPotentialConstants;
        d_cellNodeIdMapNonLocalToLocalDevice = d_cellNodeIdMapNonLocalToLocal;

        if (d_isMallocCalled)
          {
            free(h_d_A);
            free(h_d_B);
            free(h_d_C);
            CUDACHECK(hipFree(d_A));
            CUDACHECK(hipFree(d_B));
            CUDACHECK(hipFree(d_C));
          }
        h_d_A = (dataTypes::numberGPU **)malloc(d_totalNonlocalElems *
                                                sizeof(dataTypes::numberGPU *));
        h_d_B = (dataTypes::numberGPU **)malloc(d_totalNonlocalElems *
                                                sizeof(dataTypes::numberGPU *));
        h_d_C = (dataTypes::numberGPU **)malloc(d_totalNonlocalElems *
                                                sizeof(dataTypes::numberGPU *));

        for (unsigned int i = 0; i < d_totalNonlocalElems; i++)
          {
            h_d_A[i] =
              reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
                &d_cellWaveFunctionMatrix[d_nonlocalElemIdToLocalElemIdMap[i] *
                                          numberWaveFunctions *
                                          d_numberNodesPerElement]));
            h_d_C[i] =
              reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
                &d_projectorKetTimesVectorAllCellsDevice
                  [i * numberWaveFunctions * d_maxSingleAtomPseudoWfc]));
          }

        hipMalloc((void **)&d_A,
                   d_totalNonlocalElems * sizeof(dataTypes::numberGPU *));
        hipMalloc((void **)&d_B,
                   d_totalNonlocalElems * sizeof(dataTypes::numberGPU *));
        hipMalloc((void **)&d_C,
                   d_totalNonlocalElems * sizeof(dataTypes::numberGPU *));

        hipMemcpy(d_A,
                   h_d_A,
                   d_totalNonlocalElems * sizeof(dataTypes::number *),
                   hipMemcpyHostToDevice);
        hipMemcpy(d_C,
                   h_d_C,
                   d_totalNonlocalElems * sizeof(dataTypes::number *),
                   hipMemcpyHostToDevice);

        d_isMallocCalled = true;
      }

    hipMemGetInfo(&free_t, &total_t);
    if (dftParameters::verbosity >= 2)
      pcout << "free mem after reinit allocations: " << free_t
            << ", total mem: " << total_t << std::endl;
  }

  //
  // compute mass Vector
  //
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::computeMassVector(
    const dealii::DoFHandler<3> &            dofHandler,
    const dealii::AffineConstraints<double> &constraintMatrix,
    distributedCPUVec<double> &              sqrtMassVec,
    distributedCPUVec<double> &              invSqrtMassVec)
  {
    computing_timer.enter_subsection(
      "kohnShamDFTOperatorCUDAClass Mass assembly");
    invSqrtMassVec = 0.0;
    sqrtMassVec    = 0.0;

    QGaussLobatto<3>   quadrature(FEOrder + 1);
    FEValues<3>        fe_values(dofHandler.get_fe(),
                          quadrature,
                          update_values | update_JxW_values);
    const unsigned int dofs_per_cell   = (dofHandler.get_fe()).dofs_per_cell;
    const unsigned int num_quad_points = quadrature.size();
    Vector<double>     massVectorLocal(dofs_per_cell);
    std::vector<dealii::types::global_dof_index> local_dof_indices(
      dofs_per_cell);


    //
    // parallel loop over all elements
    //
    typename DoFHandler<3>::active_cell_iterator cell =
                                                   dofHandler.begin_active(),
                                                 endc = dofHandler.end();
    for (; cell != endc; ++cell)
      if (cell->is_locally_owned())
        {
          // compute values for the current element
          fe_values.reinit(cell);
          massVectorLocal = 0.0;
          for (unsigned int i = 0; i < dofs_per_cell; ++i)
            for (unsigned int q_point = 0; q_point < num_quad_points; ++q_point)
              massVectorLocal(i) += fe_values.shape_value(i, q_point) *
                                    fe_values.shape_value(i, q_point) *
                                    fe_values.JxW(q_point);

          cell->get_dof_indices(local_dof_indices);
          constraintMatrix.distribute_local_to_global(massVectorLocal,
                                                      local_dof_indices,
                                                      invSqrtMassVec);
        }

    invSqrtMassVec.compress(VectorOperation::add);


    for (dealii::types::global_dof_index i = 0; i < invSqrtMassVec.size(); ++i)
      if (invSqrtMassVec.in_local_range(i) &&
          !constraintMatrix.is_constrained(i))
        {
          if (std::abs(invSqrtMassVec(i)) > 1.0e-15)
            {
              sqrtMassVec(i)    = std::sqrt(invSqrtMassVec(i));
              invSqrtMassVec(i) = 1.0 / std::sqrt(invSqrtMassVec(i));
            }
          AssertThrow(
            !std::isnan(invSqrtMassVec(i)),
            ExcMessage(
              "Value of inverse square root of mass matrix on the unconstrained node is undefined"));
        }

    invSqrtMassVec.compress(VectorOperation::insert);
    sqrtMassVec.compress(VectorOperation::insert);

    invSqrtMassVec.update_ghost_values();
    sqrtMassVec.update_ghost_values();

    const unsigned int numberLocalDofs = invSqrtMassVec.local_size();
    const unsigned int numberGhostDofs =
      invSqrtMassVec.get_partitioner()->n_ghost_indices();
    d_invSqrtMassVectorDevice.clear();
    d_sqrtMassVectorDevice.clear();
    d_invSqrtMassVectorDevice.resize(numberLocalDofs + numberGhostDofs);
    d_sqrtMassVectorDevice.resize(numberLocalDofs + numberGhostDofs);

    hipMemcpy(thrust::raw_pointer_cast(&d_invSqrtMassVectorDevice[0]),
               invSqrtMassVec.begin(),
               (numberLocalDofs + numberGhostDofs) * sizeof(double),
               hipMemcpyHostToDevice);

    hipMemcpy(thrust::raw_pointer_cast(&d_sqrtMassVectorDevice[0]),
               sqrtMassVec.begin(),
               (numberLocalDofs + numberGhostDofs) * sizeof(double),
               hipMemcpyHostToDevice);

    computing_timer.leave_subsection(
      "kohnShamDFTOperatorCUDAClass Mass assembly");
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::reinitkPointSpinIndex(
    const unsigned int kPointIndex,
    const unsigned int spinIndex)
  {
    d_kPointIndex = kPointIndex;
    d_spinIndex   = spinIndex;

    if (dftParameters::isPseudopotential)
      {
        for (unsigned int i = 0; i < d_totalNonlocalElems; i++)
          {
            h_d_B[i] =
              reinterpret_cast<dataTypes::numberGPU *>(thrust::raw_pointer_cast(
                &d_cellHamiltonianMatrixNonLocalFlattenedConjugateDevice
                  [d_kPointIndex * d_totalNonlocalElems *
                     d_numberNodesPerElement * d_maxSingleAtomPseudoWfc +
                   i * d_numberNodesPerElement * d_maxSingleAtomPseudoWfc]));
          }

        hipMemcpy(d_B,
                   h_d_B,
                   d_totalNonlocalElems * sizeof(dataTypes::number *),
                   hipMemcpyHostToDevice);
      }
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::computeVEff(
    const std::map<dealii::CellId, std::vector<double>> *rhoValues,
    const std::map<dealii::CellId, std::vector<double>> &phiValues,
    const std::map<dealii::CellId, std::vector<double>> &externalPotCorrValues,
    const std::map<dealii::CellId, std::vector<double>> &rhoCoreValues,
    const unsigned int externalPotCorrQuadratureId)
  {
    const unsigned int n_cells = dftPtr->matrix_free_data.n_macro_cells();
    const unsigned int totalLocallyOwnedCells =
      dftPtr->matrix_free_data.n_physical_cells();

    const Quadrature<3> &quadrature_formula =
      dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId);
    const unsigned int numberQuadraturePoints = quadrature_formula.size();

    d_vEff.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_vEffJxW.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    typename dealii::DoFHandler<3>::active_cell_iterator cellPtr =
      dftPtr->matrix_free_data.get_dof_handler().begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtr =
      dftPtr->matrix_free_data.get_dof_handler().end();
    unsigned int iElemCount = 0;

    std::vector<double> exchangePotentialVal(numberQuadraturePoints);
    std::vector<double> corrPotentialVal(numberQuadraturePoints);
    for (; cellPtr != endcPtr; ++cellPtr)
      if (cellPtr->is_locally_owned())
        {
          std::vector<double> densityValue =
            (*rhoValues).find(cellPtr->id())->second;

          if (dftParameters::nonLinearCoreCorrection)
            {
              const std::vector<double> &temp2 =
                rhoCoreValues.find(cellPtr->id())->second;
              for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
                densityValue[q] += temp2[q];
            }

          const std::vector<double> &tempPhi =
            phiValues.find(cellPtr->id())->second;

          xc_lda_vxc(&(dftPtr->funcX),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &exchangePotentialVal[0]);
          xc_lda_vxc(&(dftPtr->funcC),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &corrPotentialVal[0]);

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              d_vEff[iElemCount * numberQuadraturePoints + q] =
                tempPhi[q] + exchangePotentialVal[q] + corrPotentialVal[q];

              d_vEffJxW[iElemCount * numberQuadraturePoints + q] =
                d_vEff[iElemCount * numberQuadraturePoints + q] *
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
            }

          iElemCount++;
        }

    d_vEffJxWDevice = d_vEffJxW;
    if ((dftParameters::isPseudopotential ||
         dftParameters::smearedNuclearCharges) &&
        !d_isStiffnessMatrixExternalPotCorrComputed)
      computeVEffExternalPotCorr(externalPotCorrValues,
                                 externalPotCorrQuadratureId);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::computeVEff(
    const std::map<dealii::CellId, std::vector<double>> *rhoValues,
    const std::map<dealii::CellId, std::vector<double>> *gradRhoValues,
    const std::map<dealii::CellId, std::vector<double>> &phiValues,
    const std::map<dealii::CellId, std::vector<double>> &externalPotCorrValues,
    const std::map<dealii::CellId, std::vector<double>> &rhoCoreValues,
    const std::map<dealii::CellId, std::vector<double>> &gradRhoCoreValues,
    const unsigned int externalPotCorrQuadratureId)
  {
    const unsigned int n_cells = dftPtr->matrix_free_data.n_macro_cells();
    const unsigned int totalLocallyOwnedCells =
      dftPtr->matrix_free_data.n_physical_cells();

    const Quadrature<3> &quadrature_formula =
      dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId);
    const unsigned int numberQuadraturePoints = quadrature_formula.size();


    d_vEff.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_vEffJxW.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_derExcWithSigmaTimesGradRhoJxW.resize(totalLocallyOwnedCells *
                                              numberQuadraturePoints * 3,
                                            0.0);

    typename dealii::DoFHandler<3>::active_cell_iterator cellPtr =
      dftPtr->matrix_free_data.get_dof_handler().begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtr =
      dftPtr->matrix_free_data.get_dof_handler().end();
    unsigned int iElemCount = 0;

    std::vector<double> sigmaValue(numberQuadraturePoints);
    std::vector<double> derExchEnergyWithSigmaVal(numberQuadraturePoints);
    std::vector<double> derCorrEnergyWithSigmaVal(numberQuadraturePoints);
    std::vector<double> derExchEnergyWithDensityVal(numberQuadraturePoints);
    std::vector<double> derCorrEnergyWithDensityVal(numberQuadraturePoints);

    for (; cellPtr != endcPtr; ++cellPtr)
      if (cellPtr->is_locally_owned())
        {
          std::vector<double> densityValue =
            (*rhoValues).find(cellPtr->id())->second;
          std::vector<double> gradDensityValue =
            (*gradRhoValues).find(cellPtr->id())->second;

          if (dftParameters::nonLinearCoreCorrection)
            {
              const std::vector<double> &temp2 =
                rhoCoreValues.find(cellPtr->id())->second;
              const std::vector<double> &temp3 =
                gradRhoCoreValues.find(cellPtr->id())->second;
              for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
                {
                  densityValue[q] += temp2[q];
                  gradDensityValue[3 * q + 0] += temp3[3 * q + 0];
                  gradDensityValue[3 * q + 1] += temp3[3 * q + 1];
                  gradDensityValue[3 * q + 2] += temp3[3 * q + 2];
                }
            }

          const std::vector<double> &tempPhi =
            phiValues.find(cellPtr->id())->second;

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              const double gradRhoX = gradDensityValue[3 * q + 0];
              const double gradRhoY = gradDensityValue[3 * q + 1];
              const double gradRhoZ = gradDensityValue[3 * q + 2];
              sigmaValue[q] =
                gradRhoX * gradRhoX + gradRhoY * gradRhoY + gradRhoZ * gradRhoZ;
            }

          xc_gga_vxc(&(dftPtr->funcX),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &sigmaValue[0],
                     &derExchEnergyWithDensityVal[0],
                     &derExchEnergyWithSigmaVal[0]);
          xc_gga_vxc(&(dftPtr->funcC),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &sigmaValue[0],
                     &derCorrEnergyWithDensityVal[0],
                     &derCorrEnergyWithSigmaVal[0]);

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              const double jxw =
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
              const double gradRhoX = gradDensityValue[3 * q + 0];
              const double gradRhoY = gradDensityValue[3 * q + 1];
              const double gradRhoZ = gradDensityValue[3 * q + 2];
              const double term =
                derExchEnergyWithSigmaVal[q] + derCorrEnergyWithSigmaVal[q];
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q] = term * gradRhoX * jxw;
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q + 1] =
                term * gradRhoY * jxw;
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q + 2] =
                term * gradRhoZ * jxw;
            }

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              d_vEff[iElemCount * numberQuadraturePoints + q] =
                tempPhi[q] + derExchEnergyWithDensityVal[q] +
                derCorrEnergyWithDensityVal[q];

              d_vEffJxW[iElemCount * numberQuadraturePoints + q] =
                d_vEff[iElemCount * numberQuadraturePoints + q] *
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
            }

          iElemCount++;
        }

    d_vEffJxWDevice                        = d_vEffJxW;
    d_derExcWithSigmaTimesGradRhoJxWDevice = d_derExcWithSigmaTimesGradRhoJxW;

    if ((dftParameters::isPseudopotential ||
         dftParameters::smearedNuclearCharges) &&
        !d_isStiffnessMatrixExternalPotCorrComputed)
      computeVEffExternalPotCorr(externalPotCorrValues,
                                 externalPotCorrQuadratureId);
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    computeVEffSpinPolarized(
      const std::map<dealii::CellId, std::vector<double>> *rhoValues,
      const std::map<dealii::CellId, std::vector<double>> &phiValues,
      const unsigned int                                   spinIndex,
      const std::map<dealii::CellId, std::vector<double>>
        &externalPotCorrValues,
      const std::map<dealii::CellId, std::vector<double>> &rhoCoreValues,
      const unsigned int externalPotCorrQuadratureId)
  {
    const unsigned int n_cells = dftPtr->matrix_free_data.n_macro_cells();
    const unsigned int totalLocallyOwnedCells =
      dftPtr->matrix_free_data.n_physical_cells();

    const Quadrature<3> &quadrature_formula =
      dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId);
    const unsigned int numberQuadraturePoints = quadrature_formula.size();

    d_vEff.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_vEffJxW.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    typename dealii::DoFHandler<3>::active_cell_iterator cellPtr =
      dftPtr->matrix_free_data.get_dof_handler().begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtr =
      dftPtr->matrix_free_data.get_dof_handler().end();
    unsigned int iElemCount = 0;

    std::vector<double> exchangePotentialVal(2 * numberQuadraturePoints);
    std::vector<double> corrPotentialVal(2 * numberQuadraturePoints);
    for (; cellPtr != endcPtr; ++cellPtr)
      if (cellPtr->is_locally_owned())
        {
          std::vector<double> densityValue =
            (*rhoValues).find(cellPtr->id())->second;
          const std::vector<double> &tempPhi =
            phiValues.find(cellPtr->id())->second;

          if (dftParameters::nonLinearCoreCorrection)
            {
              const std::vector<double> &temp2 =
                rhoCoreValues.find(cellPtr->id())->second;
              for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
                {
                  densityValue[2 * q] += temp2[q] / 2.0;
                  densityValue[2 * q + 1] += temp2[q] / 2.0;
                }
            }

          xc_lda_vxc(&(dftPtr->funcX),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &exchangePotentialVal[0]);
          xc_lda_vxc(&(dftPtr->funcC),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &corrPotentialVal[0]);

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              d_vEff[iElemCount * numberQuadraturePoints + q] =
                tempPhi[q] + exchangePotentialVal[2 * q + spinIndex] +
                corrPotentialVal[2 * q + spinIndex];

              d_vEffJxW[iElemCount * numberQuadraturePoints + q] =
                d_vEff[iElemCount * numberQuadraturePoints + q] *
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
            }

          iElemCount++;
        }

    d_vEffJxWDevice = d_vEffJxW;

    if ((dftParameters::isPseudopotential ||
         dftParameters::smearedNuclearCharges) &&
        !d_isStiffnessMatrixExternalPotCorrComputed)
      computeVEffExternalPotCorr(externalPotCorrValues,
                                 externalPotCorrQuadratureId);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    computeVEffSpinPolarized(
      const std::map<dealii::CellId, std::vector<double>> *rhoValues,
      const std::map<dealii::CellId, std::vector<double>> *gradRhoValues,
      const std::map<dealii::CellId, std::vector<double>> &phiValues,
      const unsigned int                                   spinIndex,
      const std::map<dealii::CellId, std::vector<double>>
        &externalPotCorrValues,
      const std::map<dealii::CellId, std::vector<double>> &rhoCoreValues,
      const std::map<dealii::CellId, std::vector<double>> &gradRhoCoreValues,
      const unsigned int externalPotCorrQuadratureId)
  {
    const unsigned int n_cells = dftPtr->matrix_free_data.n_macro_cells();
    const unsigned int totalLocallyOwnedCells =
      dftPtr->matrix_free_data.n_physical_cells();

    const Quadrature<3> &quadrature_formula =
      dftPtr->matrix_free_data.get_quadrature(dftPtr->d_densityQuadratureId);
    const unsigned int numberQuadraturePoints = quadrature_formula.size();

    d_vEff.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_vEffJxW.resize(totalLocallyOwnedCells * numberQuadraturePoints, 0.0);
    d_derExcWithSigmaTimesGradRhoJxW.resize(totalLocallyOwnedCells *
                                              numberQuadraturePoints * 3,
                                            0.0);

    typename dealii::DoFHandler<3>::active_cell_iterator cellPtr =
      dftPtr->matrix_free_data.get_dof_handler().begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtr =
      dftPtr->matrix_free_data.get_dof_handler().end();
    unsigned int iElemCount = 0;

    std::vector<double> sigmaValue(3 * numberQuadraturePoints);
    std::vector<double> derExchEnergyWithSigmaVal(3 * numberQuadraturePoints);
    std::vector<double> derCorrEnergyWithSigmaVal(3 * numberQuadraturePoints);
    std::vector<double> derExchEnergyWithDensityVal(2 * numberQuadraturePoints);
    std::vector<double> derCorrEnergyWithDensityVal(2 * numberQuadraturePoints);

    for (; cellPtr != endcPtr; ++cellPtr)
      if (cellPtr->is_locally_owned())
        {
          std::vector<double> densityValue =
            (*rhoValues).find(cellPtr->id())->second;
          std::vector<double> gradDensityValue =
            (*gradRhoValues).find(cellPtr->id())->second;
          const std::vector<double> &tempPhi =
            phiValues.find(cellPtr->id())->second;


          if (dftParameters::nonLinearCoreCorrection)
            {
              const std::vector<double> &temp2 =
                rhoCoreValues.find(cellPtr->id())->second;
              const std::vector<double> &temp3 =
                gradRhoCoreValues.find(cellPtr->id())->second;
              for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
                {
                  densityValue[2 * q] += temp2[q] / 2.0;
                  densityValue[2 * q + 1] += temp2[q] / 2.0;
                  gradDensityValue[6 * q + 0] += temp3[3 * q + 0] / 2.0;
                  gradDensityValue[6 * q + 1] += temp3[3 * q + 1] / 2.0;
                  gradDensityValue[6 * q + 2] += temp3[3 * q + 2] / 2.0;
                  gradDensityValue[6 * q + 3] += temp3[3 * q + 0] / 2.0;
                  gradDensityValue[6 * q + 4] += temp3[3 * q + 1] / 2.0;
                  gradDensityValue[6 * q + 5] += temp3[3 * q + 2] / 2.0;
                }
            }

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              double gradRhoX1 = gradDensityValue[6 * q + 0];
              double gradRhoY1 = gradDensityValue[6 * q + 1];
              double gradRhoZ1 = gradDensityValue[6 * q + 2];
              double gradRhoX2 = gradDensityValue[6 * q + 3];
              double gradRhoY2 = gradDensityValue[6 * q + 4];
              double gradRhoZ2 = gradDensityValue[6 * q + 5];
              //
              sigmaValue[3 * q + 0] = gradRhoX1 * gradRhoX1 +
                                      gradRhoY1 * gradRhoY1 +
                                      gradRhoZ1 * gradRhoZ1;
              sigmaValue[3 * q + 1] = gradRhoX1 * gradRhoX2 +
                                      gradRhoY1 * gradRhoY2 +
                                      gradRhoZ1 * gradRhoZ2;
              sigmaValue[3 * q + 2] = gradRhoX2 * gradRhoX2 +
                                      gradRhoY2 * gradRhoY2 +
                                      gradRhoZ2 * gradRhoZ2;
            }

          xc_gga_vxc(&(dftPtr->funcX),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &sigmaValue[0],
                     &derExchEnergyWithDensityVal[0],
                     &derExchEnergyWithSigmaVal[0]);
          xc_gga_vxc(&(dftPtr->funcC),
                     numberQuadraturePoints,
                     &densityValue[0],
                     &sigmaValue[0],
                     &derCorrEnergyWithDensityVal[0],
                     &derCorrEnergyWithSigmaVal[0]);

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              const double jxw =
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
              const double gradRhoX =
                gradDensityValue[6 * q + 0 + 3 * spinIndex];
              const double gradRhoY =
                gradDensityValue[6 * q + 1 + 3 * spinIndex];
              const double gradRhoZ =
                gradDensityValue[6 * q + 2 + 3 * spinIndex];
              const double gradRhoOtherX =
                gradDensityValue[6 * q + 0 + 3 * (1 - spinIndex)];
              const double gradRhoOtherY =
                gradDensityValue[6 * q + 1 + 3 * (1 - spinIndex)];
              const double gradRhoOtherZ =
                gradDensityValue[6 * q + 2 + 3 * (1 - spinIndex)];
              const double term =
                derExchEnergyWithSigmaVal[3 * q + 2 * spinIndex] +
                derCorrEnergyWithSigmaVal[3 * q + 2 * spinIndex];
              const double termOff = derExchEnergyWithSigmaVal[3 * q + 1] +
                                     derCorrEnergyWithSigmaVal[3 * q + 1];
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q] =
                (term * gradRhoX + 0.5 * termOff * gradRhoOtherX) * jxw;
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q + 1] =
                (term * gradRhoY + 0.5 * termOff * gradRhoOtherY) * jxw;
              d_derExcWithSigmaTimesGradRhoJxW[iElemCount *
                                                 numberQuadraturePoints * 3 +
                                               3 * q + 2] =
                (term * gradRhoZ + 0.5 * termOff * gradRhoOtherZ) * jxw;
            }

          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            {
              d_vEff[iElemCount * numberQuadraturePoints + q] =
                tempPhi[q] + derExchEnergyWithDensityVal[2 * q + spinIndex] +
                derCorrEnergyWithDensityVal[2 * q + spinIndex];

              d_vEffJxW[iElemCount * numberQuadraturePoints + q] =
                d_vEff[iElemCount * numberQuadraturePoints + q] *
                d_cellJxWValues[iElemCount * numberQuadraturePoints + q];
            }

          iElemCount++;
        }

    d_vEffJxWDevice                        = d_vEffJxW;
    d_derExcWithSigmaTimesGradRhoJxWDevice = d_derExcWithSigmaTimesGradRhoJxW;

    if ((dftParameters::isPseudopotential ||
         dftParameters::smearedNuclearCharges) &&
        !d_isStiffnessMatrixExternalPotCorrComputed)
      computeVEffExternalPotCorr(externalPotCorrValues,
                                 externalPotCorrQuadratureId);
  }

  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    computeVEffExternalPotCorr(
      const std::map<dealii::CellId, std::vector<double>>
        &                externalPotCorrValues,
      const unsigned int externalPotCorrQuadratureId)
  {
    d_externalPotCorrQuadratureId = externalPotCorrQuadratureId;
    const unsigned int numberPhysicalCells =
      dftPtr->matrix_free_data.n_physical_cells();
    const int numberQuadraturePoints =
      dftPtr->matrix_free_data.get_quadrature(externalPotCorrQuadratureId)
        .size();
    FEValues<3> feValues(dftPtr->matrix_free_data.get_dof_handler().get_fe(),
                         dftPtr->matrix_free_data.get_quadrature(
                           externalPotCorrQuadratureId),
                         update_JxW_values);
    d_vEffExternalPotCorrJxW.resize(numberPhysicalCells *
                                    numberQuadraturePoints);


    typename dealii::DoFHandler<3>::active_cell_iterator cellPtr =
      dftPtr->matrix_free_data.get_dof_handler().begin_active();
    typename dealii::DoFHandler<3>::active_cell_iterator endcPtr =
      dftPtr->matrix_free_data.get_dof_handler().end();

    unsigned int iElem = 0;
    for (; cellPtr != endcPtr; ++cellPtr)
      if (cellPtr->is_locally_owned())
        {
          feValues.reinit(cellPtr);
          const std::vector<double> &temp =
            externalPotCorrValues.find(cellPtr->id())->second;
          for (unsigned int q = 0; q < numberQuadraturePoints; ++q)
            d_vEffExternalPotCorrJxW[iElem * numberQuadraturePoints + q] =
              temp[q] * feValues.JxW(q);

          iElem++;
        }

    d_vEffExternalPotCorrJxWDevice = d_vEffExternalPotCorrJxW;
  }


  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::HX(
    distributedGPUVec<dataTypes::numberGPU> &    src,
    distributedGPUVec<dataTypes::numberFP32GPU> &tempFloatArray,
    distributedGPUVec<dataTypes::numberGPU> &    projectorKetTimesVector,
    const unsigned int                           localVectorSize,
    const unsigned int                           numberWaveFunctions,
    const bool                                   scaleFlag,
    const double                                 scalar,
    distributedGPUVec<dataTypes::numberGPU> &    dst,
    const bool                                   doUnscalingSrc,
    const bool                                   singlePrecCommun)
  {
    const unsigned int n_ghosts =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->n_ghost_indices();
    const unsigned int localSize =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->local_size();
    const unsigned int totalSize = localSize + n_ghosts;
    //
    // scale src vector with M^{-1/2}
    //
    scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                      256>>>(numberWaveFunctions,
                             localVectorSize,
                             scalar,
                             src.begin(),
                             thrust::raw_pointer_cast(
                               &d_invSqrtMassVectorDevice[0]));

    if (scaleFlag)
      {
        scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                          256>>>(numberWaveFunctions,
                                 localVectorSize,
                                 1.0,
                                 dst.begin(),
                                 thrust::raw_pointer_cast(
                                   &d_sqrtMassVectorDevice[0]));
      }


    if (singlePrecCommun)
      {
        convDoubleArrToFloatArr<<<(numberWaveFunctions + 255) / 256 * localSize,
                                  256>>>(numberWaveFunctions * localSize,
                                         src.begin(),
                                         tempFloatArray.begin());
        tempFloatArray.updateGhostValues();

        if (n_ghosts != 0)
          convFloatArrToDoubleArr<<<
            (numberWaveFunctions + 255) / 256 * n_ghosts,
            256>>>(numberWaveFunctions * n_ghosts,
                   tempFloatArray.begin() + localSize * numberWaveFunctions,
                   src.begin() + localSize * numberWaveFunctions);
      }
    else
      {
        src.updateGhostValues();
      }
    getOverloadedConstraintMatrix()->distribute(src, numberWaveFunctions);

    computeLocalHamiltonianTimesX(src.begin(),
                                  numberWaveFunctions,
                                  dst.begin());

    // H^{nloc}*M^{-1/2}*X
    if (dftParameters::isPseudopotential &&
        dftPtr->d_nonLocalAtomGlobalChargeIds.size() > 0)
      {
        computeNonLocalHamiltonianTimesX(src.begin(),
                                         projectorKetTimesVector,
                                         numberWaveFunctions,
                                         dst.begin());
      }

    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst,
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]),
        numberWaveFunctions);
    else
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst, numberWaveFunctions);


    src.zeroOutGhosts();
    if (singlePrecCommun)
      {
        convDoubleArrToFloatArr<<<(numberWaveFunctions + 255) / 256 * totalSize,
                                  256>>>(numberWaveFunctions * totalSize,
                                         dst.begin(),
                                         tempFloatArray.begin());
        tempFloatArray.compressAdd();

        // copy locally owned processor boundary nodes only to dst vector
        copyFloatArrToDoubleArrLocallyOwned<<<
          (numberWaveFunctions + 255) / 256 * localSize,
          256>>>(numberWaveFunctions,
                 localSize,
                 tempFloatArray.begin(),
                 thrust::raw_pointer_cast(
                   &d_locallyOwnedProcBoundaryNodesVectorDevice[0]),
                 dst.begin());

        dst.zeroOutGhosts();
      }
    else
      {
        dst.compressAdd();
      }

    //
    // M^{-1/2}*H*M^{-1/2}*X
    //
    scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                      256>>>(numberWaveFunctions,
                             localVectorSize,
                             1.0,
                             dst.begin(),
                             thrust::raw_pointer_cast(
                               &d_invSqrtMassVectorDevice[0]));


    //
    // unscale src M^{1/2}*X
    //
    if (doUnscalingSrc)
      scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                        256>>>(numberWaveFunctions,
                               localVectorSize,
                               1.0 / scalar,
                               src.begin(),
                               thrust::raw_pointer_cast(
                                 &d_sqrtMassVectorDevice[0]));
  }



  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::HX(
    distributedGPUVec<dataTypes::numberGPU> &src,
    distributedGPUVec<dataTypes::numberGPU> &projectorKetTimesVector,
    const unsigned int                       localVectorSize,
    const unsigned int                       numberWaveFunctions,
    const bool                               scaleFlag,
    const double                             scalar,
    distributedGPUVec<dataTypes::numberGPU> &dst,
    const bool                               doUnscalingSrc)
  {
    const unsigned int n_ghosts =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->n_ghost_indices();
    const unsigned int localSize =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->local_size();
    const unsigned int totalSize = localSize + n_ghosts;
    //
    // scale src vector with M^{-1/2}
    //
    scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                      256>>>(numberWaveFunctions,
                             localVectorSize,
                             scalar,
                             src.begin(),
                             thrust::raw_pointer_cast(
                               &d_invSqrtMassVectorDevice[0]));

    if (scaleFlag)
      {
        scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                          256>>>(numberWaveFunctions,
                                 localVectorSize,
                                 1.0,
                                 dst.begin(),
                                 thrust::raw_pointer_cast(
                                   &d_sqrtMassVectorDevice[0]));
      }


    src.updateGhostValues();
    getOverloadedConstraintMatrix()->distribute(src, numberWaveFunctions);

    computeLocalHamiltonianTimesX(src.begin(),
                                  numberWaveFunctions,
                                  dst.begin());

    // H^{nloc}*M^{-1/2}*X
    if (dftParameters::isPseudopotential &&
        dftPtr->d_nonLocalAtomGlobalChargeIds.size() > 0)
      {
        computeNonLocalHamiltonianTimesX(src.begin(),
                                         projectorKetTimesVector,
                                         numberWaveFunctions,
                                         dst.begin());
      }

    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst,
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]),
        numberWaveFunctions);
    else
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst, numberWaveFunctions);


    src.zeroOutGhosts();
    dst.compressAdd();

    //
    // M^{-1/2}*H*M^{-1/2}*X
    //
    scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                      256>>>(numberWaveFunctions,
                             localVectorSize,
                             1.0,
                             dst.begin(),
                             thrust::raw_pointer_cast(
                               &d_invSqrtMassVectorDevice[0]));


    //
    // unscale src M^{1/2}*X
    //
    if (doUnscalingSrc)
      scaleCUDAKernel<<<(numberWaveFunctions + 255) / 256 * localVectorSize,
                        256>>>(numberWaveFunctions,
                               localVectorSize,
                               1.0 / scalar,
                               src.begin(),
                               thrust::raw_pointer_cast(
                                 &d_sqrtMassVectorDevice[0]));
  }


  // computePart1 and computePart2 are flags used by chebyshevFilter function to
  // perform overlap of computation and communication. When either computePart1
  // or computePart1 flags are set to true all communication calls are skipped
  // as they are directly called in chebyshevFilter. Only either of computePart1
  // or computePart2 can be set to true at one time. When computePart1 is set to
  // true distrubute, computeLocalHamiltonianTimesX, and first compute part of
  // nonlocalHX are performed before the control returns back to
  // chebyshevFilter. When computePart2 is set to true, the computations in
  // computePart1 are skipped and only computations performed are: second
  // compute part of nonlocalHX, assembly (only local processor), and
  // distribute_slave_to_master.
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::HXCheby(
    distributedGPUVec<dataTypes::numberGPU> &    src,
    distributedGPUVec<dataTypes::numberFP32GPU> &tempFloatArray,
    distributedGPUVec<dataTypes::numberGPU> &    projectorKetTimesVector,
    const unsigned int                           localVectorSize,
    const unsigned int                           numberWaveFunctions,
    distributedGPUVec<dataTypes::numberGPU> &    dst,
    bool                                         chebMixedPrec,
    bool                                         computePart1,
    bool                                         computePart2)
  {
    const unsigned int n_ghosts =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->n_ghost_indices();
    const unsigned int localSize =
      dftPtr->matrix_free_data
        .get_vector_partitioner(dftPtr->d_densityDofHandlerIndex)
        ->local_size();
    const unsigned int totalSize = localSize + n_ghosts;

    if (!(computePart1 || computePart2))
      {
        if (chebMixedPrec)
          {
            convDoubleArrToFloatArr<<<(numberWaveFunctions + 255) / 256 *
                                        localSize,
                                      256>>>(numberWaveFunctions * localSize,
                                             src.begin(),
                                             tempFloatArray.begin());
            tempFloatArray.updateGhostValues();

            if (n_ghosts != 0)
              convFloatArrToDoubleArr<<<
                (numberWaveFunctions + 255) / 256 * n_ghosts,
                256>>>(numberWaveFunctions * n_ghosts,
                       tempFloatArray.begin() + localSize * numberWaveFunctions,
                       src.begin() + localSize * numberWaveFunctions);
          }
        else
          {
            src.updateGhostValues();
          }
      }

    if (!computePart2)
      getOverloadedConstraintMatrix()->distribute(src, numberWaveFunctions);


    if (!computePart2)
      computeLocalHamiltonianTimesX(src.begin(),
                                    numberWaveFunctions,
                                    dst.begin());


    // H^{nloc}*M^{-1/2}*X
    if (dftParameters::isPseudopotential &&
        dftPtr->d_nonLocalAtomGlobalChargeIds.size() > 0)
      {
        computeNonLocalHamiltonianTimesX(src.begin(),
                                         projectorKetTimesVector,
                                         numberWaveFunctions,
                                         dst.begin(),
                                         computePart2,
                                         computePart1);
      }

    if (computePart1)
      return;


    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst,
        thrust::raw_pointer_cast(&d_tempRealVec[0]),
        thrust::raw_pointer_cast(&d_tempImagVec[0]),
        numberWaveFunctions);
    else
      getOverloadedConstraintMatrix()->distribute_slave_to_master(
        dst, numberWaveFunctions);

    if (computePart2)
      return;

    src.zeroOutGhosts();

    if (chebMixedPrec)
      {
        convDoubleArrToFloatArr<<<(numberWaveFunctions + 255) / 256 * totalSize,
                                  256>>>(numberWaveFunctions * totalSize,
                                         dst.begin(),
                                         tempFloatArray.begin());
        tempFloatArray.compressAdd();

        // copy locally owned processor boundary nodes only to dst vector
        copyFloatArrToDoubleArrLocallyOwned<<<
          (numberWaveFunctions + 255) / 256 * localSize,
          256>>>(numberWaveFunctions,
                 localSize,
                 tempFloatArray.begin(),
                 thrust::raw_pointer_cast(
                   &d_locallyOwnedProcBoundaryNodesVectorDevice[0]),
                 dst.begin());

        dst.zeroOutGhosts();
      }
    else
      {
        dst.compressAdd();
      }
  }


  // X^{T}*HConj*XConj
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::XtHX(
    const dataTypes::numberGPU *                     X,
    distributedGPUVec<dataTypes::numberGPU> &        XBlock,
    distributedGPUVec<dataTypes::numberGPU> &        HXBlock,
    distributedGPUVec<dataTypes::numberGPU> &        projectorKetTimesVector,
    const unsigned int                               M,
    const unsigned int                               N,
    hipblasHandle_t &                                 handle,
    const std::shared_ptr<const dftfe::ProcessGrid> &processGrid,
    dftfe::ScaLAPACKMatrix<dataTypes::number> &      projHamPar,
    GPUCCLWrapper &                                  gpucclMpiCommDomain)
  {
    std::map<unsigned int, unsigned int> globalToLocalColumnIdMap;
    std::map<unsigned int, unsigned int> globalToLocalRowIdMap;
    linearAlgebraOperations::internal::createGlobalToLocalIdMapsScaLAPACKMat(
      processGrid, projHamPar, globalToLocalRowIdMap, globalToLocalColumnIdMap);

    // band group parallelization data structures
    const unsigned int numberBandGroups =
      dealii::Utilities::MPI::n_mpi_processes(dftPtr->interBandGroupComm);
    const unsigned int bandGroupTaskId =
      dealii::Utilities::MPI::this_mpi_process(dftPtr->interBandGroupComm);
    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
    dftUtils::createBandParallelizationIndices(dftPtr->interBandGroupComm,
                                               N,
                                               bandGroupLowHighPlusOneIndices);



    const unsigned int vectorsBlockSize =
      std::min(dftParameters::wfcBlockSize, N);

    dataTypes::number *projHamBlockHost;
    hipHostMalloc((void **)&projHamBlockHost,
                   vectorsBlockSize * N * sizeof(dataTypes::number));
    std::memset(projHamBlockHost,
                0,
                vectorsBlockSize * N * sizeof(dataTypes::number));

    thrust::device_vector<dataTypes::numberThrustGPU> HXBlockFull(
      vectorsBlockSize * M, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberThrustGPU> projHamBlock(
      vectorsBlockSize * N, dataTypes::numberThrustGPU(0.0));

    for (unsigned int jvec = 0; jvec < N; jvec += vectorsBlockSize)
      {
        // Correct block dimensions if block "goes off edge of" the matrix
        const unsigned int B = std::min(vectorsBlockSize, N - jvec);

        if ((jvec + B) <=
              bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
            (jvec + B) > bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
          {
            const unsigned int chebyBlockSize =
              std::min(dftParameters::chebyWfcBlockSize, N);

            for (unsigned int k = jvec; k < jvec + B; k += chebyBlockSize)
              {
                stridedCopyToBlockKernel<<<(chebyBlockSize + 255) / 256 * M,
                                           256>>>(
                  chebyBlockSize, M, X, N, XBlock.begin(), k);

                // evaluate XBlock^{T} times H^{T} and store in HXBlock
                HXBlock.setZero();
                // thrust::fill(HXBlock.begin(),HXBlock.end(),0.0);
                const bool   scaleFlag = false;
                const double scalar    = 1.0;
                HX(XBlock,
                   projectorKetTimesVector,
                   M,
                   chebyBlockSize,
                   scaleFlag,
                   scalar,
                   HXBlock,
                   false);

                stridedCopyFromBlockKernel<<<(chebyBlockSize + 255) / 256 * M,
                                             256>>>(
                  chebyBlockSize,
                  M,
                  HXBlock.begin(),
                  B,
                  reinterpret_cast<dataTypes::numberGPU *>(
                    thrust::raw_pointer_cast(&HXBlockFull[0])),
                  k - jvec);
              }

            // Comptute local XTrunc^{T}*HConj*XConj.
            const dataTypes::number alpha = dataTypes::number(1.0),
                                    beta  = dataTypes::number(0.0);
            const unsigned int D          = N - jvec;
            cublasXgemm(
              handle,
              HIPBLAS_OP_N,
              std::is_same<dataTypes::number, std::complex<double>>::value ?
                HIPBLAS_OP_C :
                HIPBLAS_OP_T,
              D,
              B,
              M,
              reinterpret_cast<const dataTypes::numberGPU *>(&alpha),
              X + jvec,
              N,
              reinterpret_cast<const dataTypes::numberGPU *>(
                thrust::raw_pointer_cast(&HXBlockFull[0])),
              B,
              reinterpret_cast<const dataTypes::numberGPU *>(&beta),
              reinterpret_cast<dataTypes::numberGPU *>(
                thrust::raw_pointer_cast(&projHamBlock[0])),
              D);

            hipMemcpy(projHamBlockHost,
                       reinterpret_cast<dataTypes::numberGPU *>(
                         thrust::raw_pointer_cast(&projHamBlock[0])),
                       D * B * sizeof(dataTypes::numberGPU),
                       hipMemcpyDeviceToHost);


            // Sum local projHamBlock across domain decomposition processors
            MPI_Allreduce(MPI_IN_PLACE,
                          projHamBlockHost,
                          D * B,
                          dataTypes::mpi_type_id(projHamBlockHost),
                          MPI_SUM,
                          mpi_communicator);

            // Copying only the lower triangular part to the ScaLAPACK projected
            // Hamiltonian matrix
            if (processGrid->is_process_active())
              for (unsigned int j = 0; j < B; ++j)
                if (globalToLocalColumnIdMap.find(j + jvec) !=
                    globalToLocalColumnIdMap.end())
                  {
                    const unsigned int localColumnId =
                      globalToLocalColumnIdMap[j + jvec];
                    for (unsigned int i = j + jvec; i < N; ++i)
                      {
                        std::map<unsigned int, unsigned int>::iterator it =
                          globalToLocalRowIdMap.find(i);
                        if (it != globalToLocalRowIdMap.end())
                          projHamPar.local_el(it->second, localColumnId) =
                            projHamBlockHost[j * D + i - jvec];
                      }
                  }

          } // band parallelization
      }

    CUDACHECK(hipHostFree(projHamBlockHost));

    if (numberBandGroups > 1)
      {
        MPI_Barrier(dftPtr->interBandGroupComm);
        linearAlgebraOperations::internal::sumAcrossInterCommScaLAPACKMat(
          processGrid, projHamPar, dftPtr->interBandGroupComm);
      }
  }

  // X^{T}*HConj*XConj  with overlap of computation and
  // communication
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    XtHXOverlapComputeCommun(
      const dataTypes::numberGPU *                     X,
      distributedGPUVec<dataTypes::numberGPU> &        XBlock,
      distributedGPUVec<dataTypes::numberGPU> &        HXBlock,
      distributedGPUVec<dataTypes::numberGPU> &        projectorKetTimesVector,
      const unsigned int                               M,
      const unsigned int                               N,
      hipblasHandle_t &                                 handle,
      const std::shared_ptr<const dftfe::ProcessGrid> &processGrid,
      dftfe::ScaLAPACKMatrix<dataTypes::number> &      projHamPar,
      GPUCCLWrapper &                                  gpucclMpiCommDomain)
  {
    /////////////PSEUDO CODE for the implementation below for Overlapping
    /// compute and communication/////////////////
    //
    // In the algorithm below the communication and computation of two
    // consecutive blocks of wavefunctions: block i and block i+1 are
    // overlapped.
    // ----------------------------------------------------------
    // CMP denotes computuation of X^{T} times HXBlock
    // COP denotes GPU->CPU copy of X^{T} times HXBlock
    // COM denotes blocking MPI_Allreduce on X^{T}HXBlock and copy to scalapack
    // matrix
    // ----------------------------------------------------------
    // Two CUDA streams are created: compute and copy
    // CMP is performed in compute CUDA stream and COP is performed in copy CUDA
    // stream. COP for a block can only start after the CMP for that block in
    // the compute stream is completed. COM is performed for a block only after
    // COP even for that block is completed.
    //
    // In a blocked loop do:
    // 1) [CMP] Call compute on first block (edge case only for first iteration)
    // 2) Wait for CMP event for current block to be completed.
    // 3) Swap current and next block memory (all iterations except edge case)
    // 4) [COP] Call copy on current block
    // 5) [CMP] Call compute on next block
    // 6) Wait for COP event for current block to be completed
    // 7) [COM] Perform blocking MPI_Allreduce on curent block and copy to
    // scalapack matrix
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    std::map<unsigned int, unsigned int> globalToLocalColumnIdMap;
    std::map<unsigned int, unsigned int> globalToLocalRowIdMap;
    linearAlgebraOperations::internal::createGlobalToLocalIdMapsScaLAPACKMat(
      processGrid, projHamPar, globalToLocalRowIdMap, globalToLocalColumnIdMap);

    // band group parallelization data structures
    const unsigned int numberBandGroups =
      dealii::Utilities::MPI::n_mpi_processes(dftPtr->interBandGroupComm);
    const unsigned int bandGroupTaskId =
      dealii::Utilities::MPI::this_mpi_process(dftPtr->interBandGroupComm);
    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
    dftUtils::createBandParallelizationIndices(dftPtr->interBandGroupComm,
                                               N,
                                               bandGroupLowHighPlusOneIndices);



    const unsigned int vectorsBlockSize =
      std::min(dftParameters::wfcBlockSize, N);
    const unsigned int numberBlocks = N / vectorsBlockSize;

    // create separate CUDA streams for GPU->CPU copy and computation
    hipStream_t streamCompute, streamDataMove;
    CUDACHECK(hipStreamCreate(&streamCompute));
    CUDACHECK(hipStreamCreate(&streamDataMove));

    // attach cublas handle to compute stream
    hipblasSetStream(handle, streamCompute);

    // create array of compute and copy events on GPUs
    // for all the blocks. These are required for synchronization
    // between compute, copy and communication as discussed above in the
    // pseudo code
    hipEvent_t computeEvents[numberBlocks];
    hipEvent_t copyEvents[numberBlocks];

    for (int i = 0; i < numberBlocks; ++i)
      {
        CUDACHECK(hipEventCreate(&computeEvents[i]));
        CUDACHECK(hipEventCreate(&copyEvents[i]));
      }

    dataTypes::number *projHamBlockHost;
    CUDACHECK(hipHostMalloc((void **)&projHamBlockHost,
                             vectorsBlockSize * N * sizeof(dataTypes::number)));
    std::memset(projHamBlockHost,
                0,
                vectorsBlockSize * N * sizeof(dataTypes::number));

    thrust::device_vector<dataTypes::numberThrustGPU> HXBlockFull(
      vectorsBlockSize * M, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberThrustGPU> projHamBlock(
      vectorsBlockSize * N, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberThrustGPU> projHamBlockNext(
      vectorsBlockSize * N, dataTypes::numberThrustGPU(0.0));

    dataTypes::numberValueType *tempReal;
    dataTypes::numberValueType *tempImag;
    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      {
        CUDACHECK(hipMalloc((void **)&tempReal,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberValueType)));
        CUDACHECK(hipMalloc((void **)&tempImag,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberValueType)));
      }

    unsigned int blockCount = 0;
    for (unsigned int jvec = 0; jvec < N; jvec += vectorsBlockSize)
      {
        // Correct block dimensions if block "goes off edge of" the matrix
        const unsigned int B = std::min(vectorsBlockSize, N - jvec);

        if ((jvec + B) <=
              bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
            (jvec + B) > bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
          {
            const unsigned int chebyBlockSize =
              std::min(dftParameters::chebyWfcBlockSize, N);

            const dataTypes::number alpha = dataTypes::number(1.0),
                                    beta  = dataTypes::number(0.0);
            const unsigned int D          = N - jvec;

            // handle edge case for the first block or the first block in the
            // band group in case of band parallelization
            if (jvec == bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
              {
                // compute HXBlockFull in an inner loop over blocks of B
                // wavefunction vectors
                for (unsigned int k = jvec; k < jvec + B; k += chebyBlockSize)
                  {
                    stridedCopyToBlockKernel<<<(chebyBlockSize + 255) / 256 * M,
                                               256>>>(
                      chebyBlockSize, M, X, N, XBlock.begin(), k);

                    // evaluate H times XBlock^{T} and store in HXBlock^{T}
                    HXBlock.setZero();
                    const bool   scaleFlag = false;
                    const double scalar    = 1.0;
                    HX(XBlock,
                       projectorKetTimesVector,
                       M,
                       chebyBlockSize,
                       scaleFlag,
                       scalar,
                       HXBlock,
                       false);

                    stridedCopyFromBlockKernel<<<
                      (chebyBlockSize + 255) / 256 * M,
                      256>>>(chebyBlockSize,
                             M,
                             HXBlock.begin(),
                             B,
                             reinterpret_cast<dataTypes::numberGPU *>(
                               thrust::raw_pointer_cast(&HXBlockFull[0])),
                             k - jvec);
                  }

                // evalute X^{T} times HXBlock
                cublasXgemm(
                  handle,
                  HIPBLAS_OP_N,
                  std::is_same<dataTypes::number, std::complex<double>>::value ?
                    HIPBLAS_OP_C :
                    HIPBLAS_OP_T,
                  D,
                  B,
                  M,
                  reinterpret_cast<const dataTypes::numberGPU *>(&alpha),
                  X + jvec,
                  N,
                  reinterpret_cast<const dataTypes::numberGPU *>(
                    thrust::raw_pointer_cast(&HXBlockFull[0])),
                  B,
                  reinterpret_cast<const dataTypes::numberGPU *>(&beta),
                  reinterpret_cast<dataTypes::numberGPU *>(
                    thrust::raw_pointer_cast(&projHamBlock[0])),
                  D);

                // record completion of compute for first block
                CUDACHECK(
                  hipEventRecord(computeEvents[blockCount], streamCompute));
              }


            // Before swap host thread needs to wait till compute on
            // currentblock is over. Since swap occurs on the null stream, any
            // future calls in the streamDataMove will only occur after both the
            // compute on currentblock and swap is over. Note that at this point
            // there is nothing queued in the streamDataMove as all previous
            // operations in that stream are over.
            if ((hipEventSynchronize(computeEvents[blockCount]) ==
                 hipSuccess) &&
                (jvec > bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId]))
              projHamBlock.swap(projHamBlockNext);

            const unsigned int jvecNew = jvec + vectorsBlockSize;
            const unsigned int DNew    = N - jvecNew;

            // start computations on the next block
            if (jvecNew <
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1])
              {
                for (unsigned int k = jvecNew; k < jvecNew + B;
                     k += chebyBlockSize)
                  {
                    stridedCopyToBlockKernel<<<(chebyBlockSize + 255) / 256 * M,
                                               256>>>(
                      chebyBlockSize, M, X, N, XBlock.begin(), k);

                    // evaluate H times XBlock^{T} and store in HXBlock^{T}
                    HXBlock.setZero();
                    const bool   scaleFlag = false;
                    const double scalar    = 1.0;
                    HX(XBlock,
                       projectorKetTimesVector,
                       M,
                       chebyBlockSize,
                       scaleFlag,
                       scalar,
                       HXBlock,
                       false);

                    stridedCopyFromBlockKernel<<<
                      (chebyBlockSize + 255) / 256 * M,
                      256>>>(chebyBlockSize,
                             M,
                             HXBlock.begin(),
                             B,
                             reinterpret_cast<dataTypes::numberGPU *>(
                               thrust::raw_pointer_cast(&HXBlockFull[0])),
                             k - jvecNew);
                  }

                // evalute X^{T} times HXBlock
                cublasXgemm(
                  handle,
                  HIPBLAS_OP_N,
                  std::is_same<dataTypes::number, std::complex<double>>::value ?
                    HIPBLAS_OP_C :
                    HIPBLAS_OP_T,
                  DNew,
                  B,
                  M,
                  reinterpret_cast<const dataTypes::numberGPU *>(&alpha),
                  X + jvecNew,
                  N,
                  reinterpret_cast<const dataTypes::numberGPU *>(
                    thrust::raw_pointer_cast(&HXBlockFull[0])),
                  B,
                  reinterpret_cast<const dataTypes::numberGPU *>(&beta),
                  reinterpret_cast<dataTypes::numberGPU *>(
                    thrust::raw_pointer_cast(&projHamBlockNext[0])),
                  DNew);

                // record completion of compute for next block
                CUDACHECK(hipEventRecord(computeEvents[blockCount + 1],
                                          streamCompute));
              }

            if (dftParameters::useGPUDirectAllReduce)
              {
                // Sum local projHamBlock across domain decomposition processors
                if (std::is_same<dataTypes::number,
                                 std::complex<double>>::value)
                  {
                    gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                      reinterpret_cast<dataTypes::numberGPU *>(
                        thrust::raw_pointer_cast(&projHamBlock[0])),
                      reinterpret_cast<dataTypes::numberGPU *>(
                        thrust::raw_pointer_cast(&projHamBlock[0])),
                      D * B,
                      tempReal,
                      tempImag,
                      streamDataMove);
                  }
                else
                  gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                    reinterpret_cast<dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&projHamBlock[0])),
                    reinterpret_cast<dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&projHamBlock[0])),
                    D * B,
                    streamDataMove);
              }

            hipMemcpyAsync(projHamBlockHost,
                            reinterpret_cast<const dataTypes::numberGPU *>(
                              thrust::raw_pointer_cast(&projHamBlock[0])),
                            D * B * sizeof(dataTypes::numberGPU),
                            hipMemcpyDeviceToHost,
                            streamDataMove);

            // record completion of GPU->CPU copy for current block
            CUDACHECK(hipEventRecord(copyEvents[blockCount], streamDataMove));

            // Check that GPU->CPU on the current block has been completed. If
            // completed, perform blocking MPI commmunication on the current
            // block and copy to ScaLAPACK matrix
            if (hipEventSynchronize(copyEvents[blockCount]) == hipSuccess)
              {
                // Sum local projHamBlock across domain decomposition processors
                if (!dftParameters::useGPUDirectAllReduce)
                  MPI_Allreduce(MPI_IN_PLACE,
                                projHamBlockHost,
                                D * B,
                                dataTypes::mpi_type_id(projHamBlockHost),
                                MPI_SUM,
                                mpi_communicator);

                // Copying only the lower triangular part to the ScaLAPACK
                // projected Hamiltonian matrix
                if (processGrid->is_process_active())
                  for (unsigned int j = 0; j < B; ++j)
                    if (globalToLocalColumnIdMap.find(j + jvec) !=
                        globalToLocalColumnIdMap.end())
                      {
                        const unsigned int localColumnId =
                          globalToLocalColumnIdMap[j + jvec];
                        for (unsigned int i = j + jvec; i < N; ++i)
                          {
                            std::map<unsigned int, unsigned int>::iterator it =
                              globalToLocalRowIdMap.find(i);
                            if (it != globalToLocalRowIdMap.end())
                              projHamPar.local_el(it->second, localColumnId) =
                                projHamBlockHost[j * D + i - jvec];
                          }
                      }
              }

          } // band parallelization
        blockCount += 1;
      }

    CUDACHECK(hipHostFree(projHamBlockHost));
    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      {
        CUDACHECK(hipFree(tempReal));
        CUDACHECK(hipFree(tempImag));
      }
    // return cublas handle to default stream
    hipblasSetStream(handle, NULL);

    for (int i = 0; i < numberBlocks; ++i)
      {
        CUDACHECK(hipEventDestroy(computeEvents[i]));
        CUDACHECK(hipEventDestroy(copyEvents[i]));
      }

    CUDACHECK(hipStreamDestroy(streamCompute));
    CUDACHECK(hipStreamDestroy(streamDataMove));

    if (numberBandGroups > 1)
      {
        MPI_Barrier(dftPtr->interBandGroupComm);
        linearAlgebraOperations::internal::sumAcrossInterCommScaLAPACKMat(
          processGrid, projHamPar, dftPtr->interBandGroupComm);
      }
  }


  // X^{T}*HConj*XConj  (Xc denotes complex conjugate)
  /////////////PSEUDO CODE for the implementation below for Overlapping compute
  /// and communication/////////////////
  //
  // In the algorithm below the communication and computation of two consecutive
  // blocks of wavefunctions: block i and block i+1 are overlapped.
  // ----------------------------------------------------------
  // CMP denotes computuation of X^{T} times HXBlock
  // COP denotes GPU->CPU copy of X^{T} times HXBlock
  // COM denotes blocking MPI_Allreduce on X^{T}HXBlock and copy to scalapack
  // matrix
  // ----------------------------------------------------------
  // Two CUDA streams are created: compute and copy
  // CMP is performed in compute CUDA stream and COP is performed in copy CUDA
  // stream. COP for a block can only start after the CMP for that block in the
  // compute stream is completed. COM is performed for a block only after COP
  // even for that block is completed.
  //
  // In a blocked loop do:
  // 1) [CMP] Call compute on first block (edge case only for first iteration)
  // 2) Wait for CMP event for current block to be completed.
  // 3) Swap current and next block memory (all iterations except edge case)
  // 4) [COP] Call copy on current block
  // 5) [CMP] Call compute on next block
  // 6) Wait for COP event for current block to be completed
  // 7) [COM] Perform blocking MPI_Allreduce on curent block and copy to
  // scalapack matrix
  /////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  template <unsigned int FEOrder, unsigned int FEOrderElectro>
  void
  kohnShamDFTOperatorCUDAClass<FEOrder, FEOrderElectro>::
    XtHXMixedPrecOverlapComputeCommun(
      const dataTypes::numberGPU *                     X,
      distributedGPUVec<dataTypes::numberGPU> &        XBlock,
      distributedGPUVec<dataTypes::numberFP32GPU> &    tempFloatBlock,
      distributedGPUVec<dataTypes::numberGPU> &        HXBlock,
      distributedGPUVec<dataTypes::numberGPU> &        projectorKetTimesVector,
      const unsigned int                               M,
      const unsigned int                               N,
      const unsigned int                               Noc,
      hipblasHandle_t &                                 handle,
      const std::shared_ptr<const dftfe::ProcessGrid> &processGrid,
      dftfe::ScaLAPACKMatrix<dataTypes::number> &      projHamPar,
      GPUCCLWrapper &                                  gpucclMpiCommDomain)
  {
    std::map<unsigned int, unsigned int> globalToLocalColumnIdMap;
    std::map<unsigned int, unsigned int> globalToLocalRowIdMap;
    linearAlgebraOperations::internal::createGlobalToLocalIdMapsScaLAPACKMat(
      processGrid, projHamPar, globalToLocalRowIdMap, globalToLocalColumnIdMap);

    // band group parallelization data structures
    const unsigned int numberBandGroups =
      dealii::Utilities::MPI::n_mpi_processes(dftPtr->interBandGroupComm);
    const unsigned int bandGroupTaskId =
      dealii::Utilities::MPI::this_mpi_process(dftPtr->interBandGroupComm);
    std::vector<unsigned int> bandGroupLowHighPlusOneIndices;
    dftUtils::createBandParallelizationIndices(dftPtr->interBandGroupComm,
                                               N,
                                               bandGroupLowHighPlusOneIndices);


    const unsigned int vectorsBlockSize =
      std::min(dftParameters::wfcBlockSize, N);

    const unsigned int numberBlocks = N / vectorsBlockSize;

    // create cuda compute and copy streams
    hipStream_t streamCompute, streamDataMove;
    CUDACHECK(hipStreamCreate(&streamCompute));
    CUDACHECK(hipStreamCreate(&streamDataMove));

    // attach cublas handle to compute stream
    hipblasSetStream(handle, streamCompute);

    // create array of compute and copy events on GPUs
    // for all the blocks. These are required for synchronization
    // between compute, copy and communication as discussed above in the
    // pseudo code
    hipEvent_t computeEvents[numberBlocks];
    hipEvent_t copyEvents[numberBlocks];

    for (int i = 0; i < numberBlocks; ++i)
      {
        CUDACHECK(hipEventCreate(&computeEvents[i]));
        CUDACHECK(hipEventCreate(&copyEvents[i]));
      }

    thrust::device_vector<dataTypes::numberFP32ThrustGPU> XFP32(
      M * N, dataTypes::numberFP32ThrustGPU(0.0));
    convDoubleArrToFloatArr<<<(N + 255) / 256 * M, 256>>>(
      N * M,
      X,
      reinterpret_cast<dataTypes::numberFP32GPU *>(
        thrust::raw_pointer_cast(&XFP32[0])));

    dataTypes::number *projHamBlockHost;
    CUDACHECK(hipHostMalloc((void **)&projHamBlockHost,
                             vectorsBlockSize * N * sizeof(dataTypes::number)));
    std::memset(projHamBlockHost,
                0,
                vectorsBlockSize * N * sizeof(dataTypes::number));

    dataTypes::numberFP32 *projHamBlockHostFP32;
    CUDACHECK(
      hipHostMalloc((void **)&projHamBlockHostFP32,
                     vectorsBlockSize * N * sizeof(dataTypes::numberFP32)));
    std::memset(projHamBlockHostFP32,
                0,
                vectorsBlockSize * N * sizeof(dataTypes::numberFP32));

    thrust::device_vector<dataTypes::numberThrustGPU> HXBlockFull(
      vectorsBlockSize * M, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberFP32ThrustGPU> HXBlockFullFP32(
      vectorsBlockSize * M, dataTypes::numberFP32ThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberThrustGPU> projHamBlock(
      vectorsBlockSize * N, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberFP32ThrustGPU> projHamBlockFP32(
      vectorsBlockSize * N, dataTypes::numberFP32ThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberThrustGPU> projHamBlockNext(
      vectorsBlockSize * N, dataTypes::numberThrustGPU(0.0));
    thrust::device_vector<dataTypes::numberFP32ThrustGPU> projHamBlockFP32Next(
      vectorsBlockSize * N, dataTypes::numberFP32ThrustGPU(0.0));

    dataTypes::numberValueType *    tempReal;
    dataTypes::numberValueType *    tempImag;
    dataTypes::numberFP32ValueType *tempRealFP32;
    dataTypes::numberFP32ValueType *tempImagFP32;
    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      {
        CUDACHECK(hipMalloc((void **)&tempReal,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberValueType)));
        CUDACHECK(hipMalloc((void **)&tempImag,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberValueType)));
        CUDACHECK(hipMalloc((void **)&tempRealFP32,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberFP32ValueType)));
        CUDACHECK(hipMalloc((void **)&tempImagFP32,
                             vectorsBlockSize * N *
                               sizeof(dataTypes::numberFP32ValueType)));
      }

    unsigned int blockCount = 0;
    for (unsigned int jvec = 0; jvec < N; jvec += vectorsBlockSize)
      {
        // Correct block dimensions if block "goes off edge of" the matrix
        const unsigned int B = std::min(vectorsBlockSize, N - jvec);

        if ((jvec + B) <=
              bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1] &&
            (jvec + B) > bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
          {
            const unsigned int chebyBlockSize =
              std::min(dftParameters::chebyWfcBlockSize, N);

            const dataTypes::number alpha         = dataTypes::number(1.0),
                                    beta          = dataTypes::number(0.0);
            const dataTypes::numberFP32 alphaFP32 = dataTypes::numberFP32(1.0),
                                        betaFP32  = dataTypes::numberFP32(0.0);
            const unsigned int D                  = N - jvec;

            // handle edge case for the first block or the first block in the
            // band group in case of band parallelization
            if (jvec == bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId])
              {
                // compute HXBlockFull or HXBlockFullFP32 in an inner loop over
                // blocks of B wavefunction vectors
                for (unsigned int k = jvec; k < jvec + B; k += chebyBlockSize)
                  {
                    stridedCopyToBlockKernel<dataTypes::numberGPU>
                      <<<(chebyBlockSize + 255) / 256 * M, 256>>>(
                        chebyBlockSize, M, X, N, XBlock.begin(), k);

                    // evaluate H times XBlock^{T} and store in HXBlock^{T}
                    HXBlock.setZero();
                    const bool   scaleFlag = false;
                    const double scalar    = 1.0;
                    if (jvec + B > Noc)
                      HX(XBlock,
                         projectorKetTimesVector,
                         M,
                         chebyBlockSize,
                         scaleFlag,
                         scalar,
                         HXBlock,
                         false);
                    else
                      HX(XBlock,
                         tempFloatBlock,
                         projectorKetTimesVector,
                         M,
                         chebyBlockSize,
                         scaleFlag,
                         scalar,
                         HXBlock,
                         false,
                         true);

                    if (jvec + B > Noc)
                      stridedCopyFromBlockKernel<dataTypes::numberGPU>
                        <<<(chebyBlockSize + 255) / 256 * M, 256>>>(
                          chebyBlockSize,
                          M,
                          HXBlock.begin(),
                          B,
                          reinterpret_cast<dataTypes::numberGPU *>(
                            thrust::raw_pointer_cast(&HXBlockFull[0])),
                          k - jvec);
                    else
                      stridedCopyFromBlockKernelFP32<<<
                        (chebyBlockSize + 255) / 256 * M,
                        256>>>(chebyBlockSize,
                               M,
                               HXBlock.begin(),
                               B,
                               reinterpret_cast<dataTypes::numberFP32GPU *>(
                                 thrust::raw_pointer_cast(&HXBlockFullFP32[0])),
                               k - jvec);
                  }

                // evaluate X^{T} times HXBlockFullConj or XFP32^{T} times
                // HXBlockFullFP32Conj
                if (jvec + B > Noc)
                  cublasXgemm(
                    handle,
                    HIPBLAS_OP_N,
                    std::is_same<dataTypes::number,
                                 std::complex<double>>::value ?
                      HIPBLAS_OP_C :
                      HIPBLAS_OP_T,
                    D,
                    B,
                    M,
                    reinterpret_cast<const dataTypes::numberGPU *>(&alpha),
                    X + jvec,
                    N,
                    reinterpret_cast<const dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&HXBlockFull[0])),
                    B,
                    reinterpret_cast<const dataTypes::numberGPU *>(&beta),
                    reinterpret_cast<dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&projHamBlock[0])),
                    D);
                else
                  cublasXgemm(
                    handle,
                    HIPBLAS_OP_N,
                    std::is_same<dataTypes::numberFP32,
                                 std::complex<float>>::value ?
                      HIPBLAS_OP_C :
                      HIPBLAS_OP_T,
                    D,
                    B,
                    M,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      &alphaFP32),
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&XFP32[0])) +
                      jvec,
                    N,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&HXBlockFullFP32[0])),
                    B,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      &betaFP32),
                    reinterpret_cast<dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                    D);

                // record completion of compute for next block
                CUDACHECK(
                  hipEventRecord(computeEvents[blockCount], streamCompute));
              }

            // Before swap host thread needs to wait till compute on
            // currentblock is over. Since swap occurs on the null stream, any
            // future calls in the streamDataMove will only occur after both the
            // compute on currentblock and swap is over. Note that at this point
            // there is nothing queued in the streamDataMove as all previous
            // operations in that stream are over.
            if ((hipEventSynchronize(computeEvents[blockCount]) ==
                 hipSuccess) &&
                (jvec > bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId]))
              {
                if (jvec + B > Noc)
                  projHamBlock.swap(projHamBlockNext);
                else
                  projHamBlockFP32.swap(projHamBlockFP32Next);
              }

            const unsigned int jvecNew = jvec + vectorsBlockSize;
            const unsigned int DNew    = N - jvecNew;

            if (jvecNew <
                bandGroupLowHighPlusOneIndices[2 * bandGroupTaskId + 1])
              {
                // compute HXBlockFull or HXBlockFullFP32 in an inner loop over
                // blocks of B wavefunction vectors
                for (unsigned int k = jvecNew; k < jvecNew + B;
                     k += chebyBlockSize)
                  {
                    stridedCopyToBlockKernel<dataTypes::numberGPU>
                      <<<(chebyBlockSize + 255) / 256 * M, 256>>>(
                        chebyBlockSize, M, X, N, XBlock.begin(), k);

                    // evaluate H times XBlock^{T} and store in HXBlock^{T}
                    HXBlock.setZero();
                    const bool   scaleFlag = false;
                    const double scalar    = 1.0;
                    if (jvecNew + B > Noc)
                      HX(XBlock,
                         projectorKetTimesVector,
                         M,
                         chebyBlockSize,
                         scaleFlag,
                         scalar,
                         HXBlock,
                         false);
                    else
                      HX(XBlock,
                         tempFloatBlock,
                         projectorKetTimesVector,
                         M,
                         chebyBlockSize,
                         scaleFlag,
                         scalar,
                         HXBlock,
                         false,
                         true);

                    if (jvecNew + B > Noc)
                      stridedCopyFromBlockKernel<<<
                        (chebyBlockSize + 255) / 256 * M,
                        256>>>(chebyBlockSize,
                               M,
                               HXBlock.begin(),
                               B,
                               reinterpret_cast<dataTypes::numberGPU *>(
                                 thrust::raw_pointer_cast(&HXBlockFull[0])),
                               k - jvecNew);
                    else
                      stridedCopyFromBlockKernelFP32<<<
                        (chebyBlockSize + 255) / 256 * M,
                        256>>>(chebyBlockSize,
                               M,
                               HXBlock.begin(),
                               B,
                               reinterpret_cast<dataTypes::numberFP32GPU *>(
                                 thrust::raw_pointer_cast(&HXBlockFullFP32[0])),
                               k - jvecNew);
                  }

                // evaluate X^{T} times HXBlockFullConj or XFP32^{T} times
                // HXBlockFullFP32Conj
                if (jvecNew + B > Noc)
                  cublasXgemm(
                    handle,
                    HIPBLAS_OP_N,
                    std::is_same<dataTypes::number,
                                 std::complex<double>>::value ?
                      HIPBLAS_OP_C :
                      HIPBLAS_OP_T,
                    DNew,
                    B,
                    M,
                    reinterpret_cast<const dataTypes::numberGPU *>(&alpha),
                    X + jvecNew,
                    N,
                    reinterpret_cast<const dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&HXBlockFull[0])),
                    B,
                    reinterpret_cast<const dataTypes::numberGPU *>(&beta),
                    reinterpret_cast<dataTypes::numberGPU *>(
                      thrust::raw_pointer_cast(&projHamBlockNext[0])),
                    DNew);
                else
                  cublasXgemm(
                    handle,
                    HIPBLAS_OP_N,
                    std::is_same<dataTypes::numberFP32,
                                 std::complex<float>>::value ?
                      HIPBLAS_OP_C :
                      HIPBLAS_OP_T,
                    DNew,
                    B,
                    M,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      &alphaFP32),
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&XFP32[0])) +
                      jvecNew,
                    N,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&HXBlockFullFP32[0])),
                    B,
                    reinterpret_cast<const dataTypes::numberFP32GPU *>(
                      &betaFP32),
                    reinterpret_cast<dataTypes::numberFP32GPU *>(
                      thrust::raw_pointer_cast(&projHamBlockFP32Next[0])),
                    DNew);

                // record completion of compute for next block
                CUDACHECK(hipEventRecord(computeEvents[blockCount + 1],
                                          streamCompute));
              }

            if (dftParameters::useGPUDirectAllReduce)
              {
                if (jvec + B > Noc)
                  {
                    if (std::is_same<dataTypes::number,
                                     std::complex<double>>::value)
                      gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                        reinterpret_cast<dataTypes::numberGPU *>(
                          thrust::raw_pointer_cast(&projHamBlock[0])),
                        reinterpret_cast<dataTypes::numberGPU *>(
                          thrust::raw_pointer_cast(&projHamBlock[0])),
                        D * B,
                        tempReal,
                        tempImag,
                        streamDataMove);
                    else
                      gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                        reinterpret_cast<dataTypes::numberGPU *>(
                          thrust::raw_pointer_cast(&projHamBlock[0])),
                        reinterpret_cast<dataTypes::numberGPU *>(
                          thrust::raw_pointer_cast(&projHamBlock[0])),
                        D * B,
                        streamDataMove);
                  }
                else
                  {
                    if (std::is_same<dataTypes::number,
                                     std::complex<double>>::value)
                      gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                        reinterpret_cast<dataTypes::numberFP32GPU *>(
                          thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                        reinterpret_cast<dataTypes::numberFP32GPU *>(
                          thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                        D * B,
                        tempRealFP32,
                        tempImagFP32,
                        streamDataMove);
                    else
                      gpucclMpiCommDomain.gpuDirectAllReduceWrapper(
                        reinterpret_cast<dataTypes::numberFP32GPU *>(
                          thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                        reinterpret_cast<dataTypes::numberFP32GPU *>(
                          thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                        D * B,
                        streamDataMove);
                  }
              }

            if (jvec + B > Noc)
              hipMemcpyAsync(projHamBlockHost,
                              reinterpret_cast<const dataTypes::numberGPU *>(
                                thrust::raw_pointer_cast(&projHamBlock[0])),
                              D * B * sizeof(dataTypes::number),
                              hipMemcpyDeviceToHost,
                              streamDataMove);
            else
              hipMemcpyAsync(
                projHamBlockHostFP32,
                reinterpret_cast<const dataTypes::numberFP32GPU *>(
                  thrust::raw_pointer_cast(&projHamBlockFP32[0])),
                D * B * sizeof(dataTypes::numberFP32),
                hipMemcpyDeviceToHost,
                streamDataMove);

            // record completion of GPU->CPU copy for current block
            CUDACHECK(hipEventRecord(copyEvents[blockCount], streamDataMove));

            // Check that GPU->CPU on the current block has been completed. If
            // completed, perform blocking MPI commmunication on the current
            // block and copy to ScaLAPACK matrix
            if (hipEventSynchronize(copyEvents[blockCount]) == hipSuccess)
              {
                if (jvec + B > Noc)
                  {
                    // Sum local projHamBlock across domain decomposition
                    // processors
                    if (!dftParameters::useGPUDirectAllReduce)
                      MPI_Allreduce(MPI_IN_PLACE,
                                    projHamBlockHost,
                                    D * B,
                                    dataTypes::mpi_type_id(projHamBlockHost),
                                    MPI_SUM,
                                    mpi_communicator);

                    // Copying only the lower triangular part to the ScaLAPACK
                    // projected Hamiltonian matrix
                    if (processGrid->is_process_active())
                      for (unsigned int j = 0; j < B; ++j)
                        if (globalToLocalColumnIdMap.find(j + jvec) !=
                            globalToLocalColumnIdMap.end())
                          {
                            const unsigned int localColumnId =
                              globalToLocalColumnIdMap[j + jvec];
                            for (unsigned int i = j + jvec; i < N; ++i)
                              {
                                std::map<unsigned int, unsigned int>::iterator
                                  it = globalToLocalRowIdMap.find(i);
                                if (it != globalToLocalRowIdMap.end())
                                  projHamPar.local_el(it->second,
                                                      localColumnId) =
                                    projHamBlockHost[j * D + i - jvec];
                              }
                          }
                  }
                else
                  {
                    // Sum local projHamBlock across domain decomposition
                    // processors
                    if (!dftParameters::useGPUDirectAllReduce)
                      MPI_Allreduce(MPI_IN_PLACE,
                                    projHamBlockHostFP32,
                                    D * B,
                                    dataTypes::mpi_type_id(
                                      projHamBlockHostFP32),
                                    MPI_SUM,
                                    mpi_communicator);

                    // Copying only the lower triangular part to the ScaLAPACK
                    // projected Hamiltonian matrix
                    if (processGrid->is_process_active())
                      for (unsigned int j = 0; j < B; ++j)
                        if (globalToLocalColumnIdMap.find(j + jvec) !=
                            globalToLocalColumnIdMap.end())
                          {
                            const unsigned int localColumnId =
                              globalToLocalColumnIdMap[j + jvec];
                            for (unsigned int i = j + jvec; i < N; ++i)
                              {
                                std::map<unsigned int, unsigned int>::iterator
                                  it = globalToLocalRowIdMap.find(i);
                                if (it != globalToLocalRowIdMap.end())
                                  projHamPar.local_el(it->second,
                                                      localColumnId) =
                                    projHamBlockHostFP32[j * D + i - jvec];
                              }
                          }
                  }
              }
          } // band parallelization
        blockCount += 1;
      }

    CUDACHECK(hipHostFree(projHamBlockHost));
    CUDACHECK(hipHostFree(projHamBlockHostFP32));
    if (std::is_same<dataTypes::number, std::complex<double>>::value)
      {
        CUDACHECK(hipFree(tempReal));
        CUDACHECK(hipFree(tempImag));
        CUDACHECK(hipFree(tempRealFP32));
        CUDACHECK(hipFree(tempImagFP32));
      }
    // return cublas handle to default stream
    hipblasSetStream(handle, NULL);

    for (int i = 0; i < numberBlocks; ++i)
      {
        CUDACHECK(hipEventDestroy(computeEvents[i]));
        CUDACHECK(hipEventDestroy(copyEvents[i]));
      }

    CUDACHECK(hipStreamDestroy(streamCompute));
    CUDACHECK(hipStreamDestroy(streamDataMove));

    if (numberBandGroups > 1)
      {
        MPI_Barrier(dftPtr->interBandGroupComm);
        linearAlgebraOperations::internal::sumAcrossInterCommScaLAPACKMat(
          processGrid, projHamPar, dftPtr->interBandGroupComm);
      }
  }

#include "computeNonLocalHamiltonianTimesXMemoryOptBatchGEMMCUDA.cu"
#include "hamiltonianMatrixCalculatorFlattenedCUDA.cu"
#include "inst.cu"
#include "matrixVectorProductImplementationsCUDA.cu"
#include "shapeFunctionDataCalculatorCUDA.cu"
} // namespace dftfe
