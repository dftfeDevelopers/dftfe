
#include <hip/hip_runtime.h>
// ---------------------------------------------------------------------
//
// Copyright (c) 2017-2022 The Regents of the University of Michigan and DFT-FE
// authors.
//
// This file is part of the DFT-FE code.
//
// The DFT-FE code is free software; you can use it, redistribute
// it, and/or modify it under the terms of the GNU Lesser General
// Public License as published by the Free Software Foundation; either
// version 2.1 of the License, or (at your option) any later version.
// The full text of the license can be found in the file LICENSE at
// the top level of the DFT-FE distribution.
//
// ---------------------------------------------------------------------
//
// @author Sambit Das, David M. Rogers
//

#if defined(DFTFE_WITH_GPU)

#  include <iostream>

#  include "gpuDirectCCLWrapper.h"
#  include "cudaHelpers.h"

#  if defined(DFTFE_WITH_NCCL)
#    include <nccl.h>
#  endif

namespace dftfe
{
#  define MPICHECK(cmd)                                                      \
    do                                                                       \
      {                                                                      \
        int e = cmd;                                                         \
        if (e != MPI_SUCCESS)                                                \
          {                                                                  \
            printf("Failed: MPI error %s:%d '%d'\n", __FILE__, __LINE__, e); \
            exit(EXIT_FAILURE);                                              \
          }                                                                  \
      }                                                                      \
    while (0)

#  if defined(DFTFE_WITH_NCCL)
#    define NCCLCHECK(cmd)                              \
      do                                                \
        {                                               \
          ncclResult_t r = cmd;                         \
          if (r != ncclSuccess)                         \
            {                                           \
              printf("Failed, NCCL error %s:%d '%s'\n", \
                     __FILE__,                          \
                     __LINE__,                          \
                     ncclGetErrorString(r));            \
              exit(EXIT_FAILURE);                       \
            }                                           \
        }                                               \
      while (0)
#  endif

  GPUCCLWrapper::GPUCCLWrapper()
    : commCreated(false)
  {}

  void
  GPUCCLWrapper::init(const MPI_Comm &mpiComm)
  {
    MPICHECK(MPI_Comm_size(mpiComm, &totalRanks));
    MPICHECK(MPI_Comm_rank(mpiComm, &myRank));
#  ifdef DFTFE_WITH_NCCL
    ncclIdPtr   = (void *)(new ncclUniqueId);
    ncclCommPtr = (void *)(new ncclComm_t);
    if (myRank == 0)
      ncclGetUniqueId((ncclUniqueId *)ncclIdPtr);
    MPICHECK(MPI_Bcast(
      ncclIdPtr, sizeof(*((ncclUniqueId *)ncclIdPtr)), MPI_BYTE, 0, mpiComm));
    NCCLCHECK(ncclCommInitRank((ncclComm_t *)ncclCommPtr,
                               totalRanks,
                               *((ncclUniqueId *)ncclIdPtr),
                               myRank));
    commCreated = true;
#  endif
  }

  GPUCCLWrapper::~GPUCCLWrapper()
  {
#  ifdef DFTFE_WITH_NCCL
    if (commCreated)
      {
        ncclCommDestroy(*((ncclComm_t *)ncclCommPtr));
        delete (ncclComm_t *)ncclCommPtr;
        delete (ncclUniqueId *)ncclIdPtr;
      }
#  endif
  }

  int
  GPUCCLWrapper::gpuDirectAllReduceWrapper(const float * send,
                                           float *       recv,
                                           int           size,
                                           cudaStream_t &stream)
  {
#  ifdef DFTFE_WITH_NCCL
    NCCLCHECK(ncclAllReduce((const void *)send,
                            (void *)recv,
                            size,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
#  endif
    return 0;
  }

  int
  GPUCCLWrapper::gpuDirectAllReduceWrapper(const double *send,
                                           double *      recv,
                                           int           size,
                                           cudaStream_t &stream)
  {
#  ifdef DFTFE_WITH_NCCL
    NCCLCHECK(ncclAllReduce((const void *)send,
                            (void *)recv,
                            size,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
#  endif
    return 0;
  }


  int
  GPUCCLWrapper::gpuDirectAllReduceWrapper(const cuDoubleComplex *send,
                                           cuDoubleComplex *      recv,
                                           int                    size,
                                           double *               tempReal,
                                           double *               tempImag,
                                           cudaStream_t &         stream)
  {
    cudaUtils::copyComplexArrToRealArrsGPU(size, send, tempReal, tempImag);
#  ifdef DFTFE_WITH_NCCL
    ncclGroupStart();
    NCCLCHECK(ncclAllReduce((const void *)tempReal,
                            (void *)tempReal,
                            size,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)tempImag,
                            (void *)tempImag,
                            size,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    ncclGroupEnd();
#  endif

    cudaUtils::copyRealArrsToComplexArrGPU(size, tempReal, tempImag, recv);
    return 0;
  }

  int
  GPUCCLWrapper::gpuDirectAllReduceWrapper(const cuFloatComplex *send,
                                           cuFloatComplex *      recv,
                                           int                   size,
                                           float *               tempReal,
                                           float *               tempImag,
                                           cudaStream_t &        stream)
  {
    cudaUtils::copyComplexArrToRealArrsGPU(size, send, tempReal, tempImag);
#  ifdef DFTFE_WITH_NCCL
    ncclGroupStart();
    NCCLCHECK(ncclAllReduce((const void *)tempReal,
                            (void *)tempReal,
                            size,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)tempImag,
                            (void *)tempImag,
                            size,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    ncclGroupEnd();
#  endif

    cudaUtils::copyRealArrsToComplexArrGPU(size, tempReal, tempImag, recv);
    return 0;
  }


  int
  GPUCCLWrapper::gpuDirectAllReduceMixedPrecGroupWrapper(const double *send1,
                                                         const float * send2,
                                                         double *      recv1,
                                                         float *       recv2,
                                                         int           size1,
                                                         int           size2,
                                                         cudaStream_t &stream)
  {
#  ifdef DFTFE_WITH_NCCL
    ncclGroupStart();
    NCCLCHECK(ncclAllReduce((const void *)send1,
                            (void *)recv1,
                            size1,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)send2,
                            (void *)recv2,
                            size2,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    ncclGroupEnd();
#  endif
    return 0;
  }

  int
  GPUCCLWrapper::gpuDirectAllReduceMixedPrecGroupWrapper(
    const cuDoubleComplex *send1,
    const cuFloatComplex * send2,
    cuDoubleComplex *      recv1,
    cuFloatComplex *       recv2,
    int                    size1,
    int                    size2,
    double *               tempReal1,
    float *                tempReal2,
    double *               tempImag1,
    float *                tempImag2,
    cudaStream_t &         stream)
  {
    cudaUtils::copyComplexArrToRealArrsGPU(size1, send1, tempReal1, tempImag1);

    cudaUtils::copyComplexArrToRealArrsGPU(size2, send2, tempReal2, tempImag2);

#  ifdef DFTFE_WITH_NCCL
    ncclGroupStart();
    NCCLCHECK(ncclAllReduce((const void *)tempReal1,
                            (void *)tempReal1,
                            size1,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)tempImag1,
                            (void *)tempImag1,
                            size1,
                            ncclDouble,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)tempReal2,
                            (void *)tempReal2,
                            size2,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    NCCLCHECK(ncclAllReduce((const void *)tempImag2,
                            (void *)tempImag2,
                            size2,
                            ncclFloat,
                            ncclSum,
                            *((ncclComm_t *)ncclCommPtr),
                            stream));
    ncclGroupEnd();
#  endif

    cudaUtils::copyRealArrsToComplexArrGPU(size1, tempReal1, tempImag1, recv1);

    cudaUtils::copyRealArrsToComplexArrGPU(size2, tempReal2, tempImag2, recv2);

    return 0;
  }

} // namespace dftfe
#endif
